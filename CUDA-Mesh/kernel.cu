#include "hip/hip_runtime.h"
#include "Device.h"
#include <math_functions.h>

/*
#define FOV_Y 43 # degrees
#define FOV_X 57

#define SCALE_Y tan((FOV_Y/2)*pi/180)
#define SCALE_X tan((FOV_X/2)*pi/180)
*/
#define SCALE_Y 0.393910475614942392
#define SCALE_X 0.542955699638436879
#define PI      3.141592653589793238
#define MIN_EIG_RATIO 1.5

#define RAD_WIN 4
#define RAD_NN 0.05
#define MIN_NN int((2*RAD_WIN+1)*0.5)

ColorPixel* dev_colorImageBuffer;
DPixel* dev_depthImageBuffer;
PointCloud* dev_pointCloudBuffer;

int	cuImageWidth = 0;
int	cuImageHeight = 0;

GLuint imagePBO = (GLuint)NULL;

__host__ void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		exit(EXIT_FAILURE); 
	}
} 



__global__ void makePointCloud(ColorPixel* colorPixels, DPixel* dPixels, int xRes, int yRes, PointCloud** pointCloud) {
	int i = (blockIdx.y*gridDim.x + blockIdx.x)*(blockDim.y*blockDim.x) + (threadIdx.y*blockDim.x) + threadIdx.x;
	int r = i / xRes;
	int c = i % xRes;

	if (dPixels[i].depth > 0.0f) {
		float u = (c - (xRes-1)/2.0f + 1) / (xRes-1); // image plane u coordinate
		float v = ((yRes-1)/2.0f - r) / (yRes-1); // image plane v coordinate
		float Z = dPixels[i].depth/1000.0f; // depth in mm
        if (Z > 0.0f) {
		    pointCloud[r][c].pos = glm::vec3(u*Z*SCALE_X, v*Z*SCALE_Y, Z); // convert uv to XYZ
        } else {
            pointCloud[r][c].pos = glm::vec3(0.0f);
        }
		pointCloud[r][c].color = glm::vec3(colorPixels[i].r, colorPixels[i].g, colorPixels[i].b); // copy over texture
	}
}

__device__ glm::vec3 normalFrom3x3Covar(glm::mat3 A) {
	// Given a (real, symmetric) 3x3 covariance matrix A, returns the eigenvector corresponding to the min eigenvalue
	// (see: http://en.wikipedia.org/wiki/Eigenvalue_algorithm#3.C3.973_matrices)
    glm::vec3 eigs;
    glm::vec3 normal = glm::vec3(0.0f);
	float p1 = pow(A[0][1], 2) + pow(A[0][2], 2) + pow(A[1][2], 2);
	if (p1 == 0) { // A is diagonal
		eigs = glm::vec3(A[0][0], A[1][1], A[2][2]);
	} else {
		float q = (A[0][0] + A[1][1] + A[2][2])/3.0f; // mean(trace(A))
		float p2 = pow(A[0][0]-q, 2) + pow(A[1][1]-q, 2) + pow(A[2][2]-q, 2) + 2*p1;
		float p = sqrt(p2/6);
		glm::mat3 B = (1/p) * (A-q*glm::mat3(1.0f));
		float r = glm::determinant(B)/2;
		// theoretically -1 <= r <= 1, but clamp in case of numeric error
		float phi;
		if (r <= -1) {
			phi = PI / 3;
		} else if (r >= 1) {
			phi = 0;
		} else {
			phi = glm::acos(r)/3;
		}
		eigs[0] = q + 2*p*glm::cos(phi);
		eigs[1] = q + 2*p*glm::cos(phi + 2*PI/3);
		eigs[2] = 3*q - eigs.x - eigs.z;
        float tmp;
        int i, eig_i;
        // sorting: swap first pair if necessary, then second pair, then first pair again
        for (i=0; i<3; i++) {
            eig_i = i%2;
            tmp = eigs[eig_i];
            eigs[eig_i] = glm::min(tmp, eigs[eig_i+1]);
            eigs[eig_i+1] = glm::max(tmp, eigs[eig_i+1]);
        }
	}
	// compute eigenvector from min eigenvalue if point cloud is sufficiently "flat"
    if (eigs[1]/eigs[0] >= MIN_EIG_RATIO) {
	  normal = glm::cross(A[0] - glm::vec3(eigs[0], 0.0f, 0.0f), A[1] - glm::vec3(0.0f, eigs[0], 0.0f));
    }
    return normal;
}

__global__ void computePointNormals(PointCloud** pointCloud, int xRes, int yRes) {
	int i = (blockIdx.y*gridDim.x + blockIdx.x)*(blockDim.y*blockDim.x) + (threadIdx.y*blockDim.x) + threadIdx.x;
	int r = i / xRes;
	int c = i % xRes;



}


//Kernel that writes the depth image to the OpenGL PBO directly.
__global__ void sendDepthImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, DPixel* depthBuffer){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<resolution.x && y<resolution.y) {

		//Cast to float for storage
		float depth = depthBuffer[index].depth;

		// Each thread writes one pixel location in the texture (textel)
		//Store depth in every component except alpha
		PBOpos[index].x = depth;
		PBOpos[index].y = depth;
		PBOpos[index].z = depth;
		PBOpos[index].w = 1.0f;
	}
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendColorImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, ColorPixel* colorBuffer){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<resolution.x && y<resolution.y){

		glm::vec3 color;
		color.r = colorBuffer[index].r/255.0f;
		color.g = colorBuffer[index].g/255.0f;
		color.b = colorBuffer[index].b/255.0f;


		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].x = color.r;
		PBOpos[index].y = color.g;
		PBOpos[index].z = color.b;
		PBOpos[index].w = 1.0f;
	}
}

__host__ void deletePBO(GLuint *pbo)
{
	if (pbo) {
		// unregister this buffer object with CUDA
		cudaGLUnregisterBufferObject(*pbo);

		glBindBuffer(GL_ARRAY_BUFFER, *pbo);
		glDeleteBuffers(1, pbo);

		*pbo = (GLuint)NULL;
	}
}


//Intialize pipeline buffers
__host__ void initCuda(int width, int height)
{
	//Allocate buffers
	hipMalloc((void**) &dev_colorImageBuffer, sizeof(ColorPixel)*width*height);
	hipMalloc((void**) &dev_depthImageBuffer, sizeof(DPixel)*width*height);
	hipMalloc((void**) &dev_pointCloudBuffer, sizeof(PointCloud)*width*height);
	cuImageWidth = width;
	cuImageHeight = height;

}

//Free all allocated buffers and close out environment
__host__ void cleanupCuda()
{
	if(imagePBO) deletePBO(&imagePBO);

	hipFree(dev_colorImageBuffer);
	hipFree(dev_depthImageBuffer);
	hipFree(dev_pointCloudBuffer);
	cuImageWidth = 0;
	cuImageHeight = 0;

	hipDeviceReset();

}


//Copies a depth image to the GPU buffer. 
//Returns false if width and height do not match buffer size set by initCuda(), true if success
__host__ bool pushDepthArrayToBuffer(DPixel* hDepthArray, int width, int height)
{
	if(width != cuImageWidth || height != cuImageHeight)
		return false;//Buffer wrong size

	hipMemcpy(dev_depthImageBuffer, hDepthArray, sizeof(DPixel)*width*height, hipMemcpyHostToDevice);
	return true;
}


//Copies a color image to the GPU buffer. 
//Returns false if width and height do not match buffer size set by initCuda(), true if success
__host__ bool pushColorArrayToBuffer(ColorPixel* hColorArray, int width, int height)
{
	if(width != cuImageWidth || height != cuImageHeight)
		return false;//Buffer wrong size

	hipMemcpy((void*)dev_colorImageBuffer, hColorArray, sizeof(ColorPixel)*width*height, hipMemcpyHostToDevice);
	return true;
}

//Converts the color and depth images currently in GPU buffers into point cloud buffer
__host__ void convertToPointCloud()
{
	//TODO: Implement

}

//Computes normals for point cloud in buffer and writes back to the point cloud buffer.
__host__ void computePointCloudNormals()
{
	//TODO: Implement

}


//Draws depth image buffer to the texture.
//Texture width and height must match the resolution of the depth image.
//Returns false if width or height does not match, true otherwise
bool drawDepthImageBufferToPBO(float4* dev_PBOpos, int texWidth, int texHeight)
{
	if(texWidth != cuImageWidth || texHeight != cuImageHeight)
		return false;

	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendDepthImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_depthImageBuffer);
	
	return true;
}

//Draws color image buffer to the texture.
//Texture width and height must match the resolution of the color image.
//Returns false if width or height does not match, true otherwise
//dev_PBOpos must be a CUDA device pointer
bool drawColorImageBufferToPBO(float4* dev_PBOpos, int texWidth, int texHeight)
{
	if(texWidth != cuImageWidth || texHeight != cuImageHeight)
		return false;

	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendColorImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_colorImageBuffer);
	
	return true;
}

//Renders the point cloud as stored in the VBO to the texture
__host__ void drawPointCloudVBOToTexture(GLuint texture, int texWidth, int texHeight /*TODO: More vizualization parameters here*/)
{
	//TODO: Implement

}

//Renders various debug information about the 2D point cloud buffer to the texture.
//Texture width and height must match the resolution of the point cloud buffer.
//Returns false if width or height does not match, true otherwise
__host__ bool drawPCBToPBO(float4* dptrPosition, float4* dptrColor, float4* dptrNormal, int mXRes, int mYRes)
{
	return false;
}
