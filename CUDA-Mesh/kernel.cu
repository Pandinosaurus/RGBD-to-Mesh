#include "hip/hip_runtime.h"
#include "Device.h"
#include ""
#include "math_functions.h"

ColorPixel* dev_colorImageBuffer;
DPixel* dev_depthImageBuffer;
PointCloud* dev_pointCloudBuffer;
PointCloud* dev_pointCloudVBO;

triangleIndecies* dev_triangulationIBO;
triangleIndecies* dev_triangulationIBOCompact;


int	cuImageWidth = 0;
int	cuImageHeight = 0;

GLuint imagePBO = (GLuint)NULL;


__host__ void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		exit(EXIT_FAILURE); 
	}
}

__global__ void makePointCloud(ColorPixel* colorPixels, DPixel* dPixels, int xRes, int yRes, PointCloud* pointCloud) {
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * xRes) + c;

	if(r < yRes && c < xRes) {
		// In range
		if (dPixels[i].depth != 0) {
			float u = (c - (xRes-1.0f)/2.0f + 1.0f) / (xRes-1.0f); // image plane u coordinate
			float v = ((yRes-1.0f)/2.0f - r) / (yRes-1.0f); // image plane v coordinate
			float Z = dPixels[i].depth/1000.0f; // depth converted to meters
			pointCloud[i].pos = glm::vec3(u*Z*SCALE_X, v*Z*SCALE_Y, -Z); // convert uv to XYZ
			pointCloud[i].color = glm::vec3(colorPixels[i].r/255.0f, colorPixels[i].g/255.0f, colorPixels[i].b/255.0f); // copy over texture
		} else {
			pointCloud[i].pos = glm::vec3(0.0f);
			pointCloud[i].color = glm::vec3(0.0f);
		}
		// Always clear normals
		pointCloud[i].normal = glm::vec3(0.0f);
	}
}

__device__ glm::vec3 normalFrom3x3Covar(glm::mat3 A) {
	// Given a (real, symmetric) 3x3 covariance matrix A, returns the eigenvector corresponding to the min eigenvalue
	// (see: http://en.wikipedia.org/wiki/Eigenvalue_algorithm#3.C3.973_matrices)
	glm::vec3 eigs;
	glm::vec3 normal = glm::vec3(0.0f);
	float p1 = pow(A[0][1], 2) + pow(A[0][2], 2) + pow(A[1][2], 2);
	if (abs(p1) < EPSILON) { // A is diagonal
		eigs = glm::vec3(A[0][0], A[1][1], A[2][2]);
	} else {
		float q = (A[0][0] + A[1][1] + A[2][2])/3.0f; // mean(trace(A))
		float p2 = pow(A[0][0]-q, 2) + pow(A[1][1]-q, 2) + pow(A[2][2]-q, 2) + 2*p1;
		float p = sqrt(p2/6);
		glm::mat3 B = (1/p) * (A-q*glm::mat3(1.0f));
		float r = glm::determinant(B)/2;
		// theoretically -1 <= r <= 1, but clamp in case of numeric error
		float phi;
		if (r <= -1) {
			phi = PI / 3;
		} else if (r >= 1) { 
			phi = 0;
		} else {
			phi = glm::acos(r)/3;
		}
		eigs[0] = q + 2*p*glm::cos(phi);
		eigs[2] = q + 2*p*glm::cos(phi + 2*PI/3);
		eigs[1] = 3*q - eigs[0] - eigs[2];
		float tmp;
		int i, eig_i;
		// sorting: swap first pair if necessary, then second pair, then first pair again
		for (i=0; i<3; i++) {
			eig_i = i%2;
			tmp = eigs[eig_i];
			eigs[eig_i] = glm::min(tmp, eigs[eig_i+1]);
			eigs[eig_i+1] = glm::max(tmp, eigs[eig_i+1]);
		}
	}
	// check if point cloud region is "flat" enough
	if (eigs[1]/eigs[0] >= MIN_EIG_RATIO) {
		normal = glm::normalize(glm::cross(A[0] - glm::vec3(eigs[0], 0.0f, 0.0f), A[1] - glm::vec3(0.0f, eigs[0], 0.0f)));
	}
	return normal;
}

/*
__global__ void computePointNormals(PointCloud* pointCloud, int xRes, int yRes) {
int r = (blockIdx.y * blockDim.y) + threadIdx.y;
int c = (blockIdx.x * blockDim.x) + threadIdx.x;
int i = (r * xRes) + c;

int N = 0; // number of nearest neighbors
glm::vec3 neighbor;
glm::vec3 center = pointCloud[i].pos;
glm::mat3 covariance = glm::mat3(0.0f);
glm::vec3 normal;
int win_r, win_c, win_i;
for (win_r = r-RAD_WIN; win_r <= r+RAD_WIN; win_r++) {
for (win_c = c-RAD_WIN; win_c <= c+RAD_WIN; win_c++) {
// exclude center from neighbor search
if (win_r != r && win_c != c) {
// check if neighbor is in frame
if (win_r >= 0 && win_r < yRes && win_c >= 0 && win_c < xRes) {
win_i = (win_r * xRes) + win_c;
neighbor = pointCloud[win_i].pos;
// check if neighbor has valid depth data
if (glm::length(neighbor) > EPSILON) {
// check if neighbor is close enough in world space
if (glm::distance(neighbor, center) < RAD_NN) {
N += 1; // valid neighbor found
glm::vec3 difference = neighbor - center;
// remember GLM is column major
covariance[0] += (difference * difference[0]);
covariance[1] += (difference * difference[1]);
covariance[2] += (difference * difference[2]);
}
}
}
}
}
}
// check if enough nearest neighbors were found
if (N >= MIN_NN) {
covariance = covariance/N; // average covariance
// compute and assign normal (0 if not "flat" enough)
normal = normalFrom3x3Covar(covariance);
// flip normal if facing away from camera
if (glm::dot(center, normal) > 0) {
normal = -normal;
}
pointCloud[i].normal = normal;
}
}
*/

__global__ void computePointNormals(PointCloud* pointCloud, int xRes, int yRes) {
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * xRes) + c;

	glm::vec3 center = pointCloud[i].pos;
	glm::vec3 neighbor;
	glm::vec3 neighbor_ortho;
	glm::vec3 normal_sum = glm::vec3(0.0f);
	glm::vec3 normal;
	float N = 0.0f;

	int win_r, win_c, win_i;
	for (win_r = -RAD_WIN; win_r <= RAD_WIN; win_r++) {
		for (win_c = -RAD_WIN; win_c <= RAD_WIN; win_c++) {
			if (r+win_r >= 0 && c+win_c >= 0 && r+win_r < yRes && c+win_c < xRes) {
				if (!(win_r == 0 & win_c == 0)) {
					neighbor = pointCloud[i+win_c+win_r*xRes].pos;
					neighbor_ortho = pointCloud[i-win_r+win_c*xRes].pos;
					if (glm::length(neighbor) > EPSILON && glm::length(neighbor_ortho) > EPSILON) {
						if (glm::distance(center, neighbor) < RAD_NN && glm::distance(center, neighbor_ortho) < RAD_NN) {
							normal = glm::normalize(glm::cross(neighbor-center, neighbor_ortho-center));
							normal_sum += (glm::dot(center, normal) > 0 ? -normal : normal);
							++N;
						}
					}
				}
			}
		}
	}
	if (N > MIN_NN) {
		pointCloud[i].normal = normal_sum / N;
	}
}

// Kernel that writes the depth image to the OpenGL PBO directly.
__global__ void sendDepthImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, DPixel* depthBuffer){

	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x) {

		// Cast to float for storage
		float depth = depthBuffer[i].depth;

		// Each thread writes one pixel location in the texture (textel)
		// Store depth in every component except alpha
		PBOpos[i].x = depth;
		PBOpos[i].y = depth;
		PBOpos[i].z = depth;
		PBOpos[i].w = 1.0f;
	}
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendColorImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, ColorPixel* colorBuffer){

	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x){

		glm::vec3 color;
		color.r = colorBuffer[i].r/255.0f;
		color.g = colorBuffer[i].g/255.0f;
		color.b = colorBuffer[i].b/255.0f;

		// Each thread writes one pixel location in the texture (textel)
		PBOpos[i].x = color.r;
		PBOpos[i].y = color.g;
		PBOpos[i].z = color.b;
		PBOpos[i].w = 1.0f;
	}
}

__global__ void sendPCBToPBOs(float4* dptrPosition, float4* dptrColor, float4* dptrNormal, glm::vec2 resolution, PointCloud* dev_pcb)
{
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x){

		PointCloud point = dev_pcb[i];

		// Each thread writes one pixel location in the texture (textel)

		dptrPosition[i].x = point.pos.x;
		dptrPosition[i].y = point.pos.y;
		dptrPosition[i].z = point.pos.z;
		dptrPosition[i].w = 1.0f;

		dptrColor[i].x = point.color.r;
		dptrColor[i].y = point.color.g;
		dptrColor[i].z = point.color.b;
		dptrColor[i].w = 1.0f;

		dptrNormal[i].x = point.normal.x;
		dptrNormal[i].y = point.normal.y;
		dptrNormal[i].z = point.normal.z;
		dptrNormal[i].w = 0.0f;
	}
}

__global__ void triangulationKernel(PointCloud* pointCloudBuffer, triangleIndecies* triangulationIBO, glm::vec2 resolution, float maxTriangleEdgeLength)
{
	//Parallel by proposed triangle. (x,y) indicates the upper left corner of triangle, z indicates which triangle this thread creates
	//  (x,y)__
	//		|\0|
	//		|1\|
	int x = (blockIdx.y * blockDim.y) + threadIdx.y;
	int y = (blockIdx.x * blockDim.x) + threadIdx.x;
	int triangleIndex = (threadIdx.z * resolution.x * resolution.y) + (y * resolution.x) + x;

	unsigned int i0, i1, i2;
	//Get pixel locations in image space
	if(triangleIndex < resolution.x*resolution.y*2){
		if(x < resolution.x-1 && y < resolution.y-1){
			//int v0x = x;
			//int v0y = y;
			i0 = x+resolution.x*y;

			int v1x = x + (1 - threadIdx.z);//z == 0?x + 1: x //next row if upper triangle
			int v1y = y + 1;//Always on next row
			i1 = v1x+resolution.x*v1y;

			int v2x = x + 1; //Always on next column
			int v2y = y + threadIdx.z;//z == 0 ? y : y + 1, next column if lower triangle
			i2 = v2x+resolution.x*v2y;

			//Pull all pixels to local memory
			glm::vec3 p0 = pointCloudBuffer[i0].pos;
			glm::vec3 p1 = pointCloudBuffer[i1].pos;
			glm::vec3 p2 = pointCloudBuffer[i2].pos;

			//If all points are non-zero and lengths within bounds
			if(	   glm::length(p0) < 0.001 
				|| glm::length(p1) < 0.001  
				|| glm::length(p2) < 0.001
				|| glm::length(p1-p0) > maxTriangleEdgeLength 
				|| glm::length(p2-p0) > maxTriangleEdgeLength
				|| glm::length(p2-p1) > maxTriangleEdgeLength)
			{
				i0 = 0;
				i1 = 0;
				i2 = 0;
			}
		}

		triangulationIBO[triangleIndex].v0 = i0;
		triangulationIBO[triangleIndex].v1 = i1;
		triangulationIBO[triangleIndex].v2 = i2;

	}
}

__host__ void deletePBO(GLuint *pbo)
{
	if (pbo) {
		// unregister this buffer object with CUDA
		cudaGLUnregisterBufferObject(*pbo);

		glBindBuffer(GL_ARRAY_BUFFER, *pbo);
		glDeleteBuffers(1, pbo);

		*pbo = (GLuint)NULL;
	}
}

// Intialize pipeline buffers
__host__ void initCuda(int width, int height)
{
	// Allocate buffers
	hipMalloc((void**) &dev_colorImageBuffer,			sizeof(ColorPixel)*width*height);
	hipMalloc((void**) &dev_depthImageBuffer,			sizeof(DPixel)*width*height);
	hipMalloc((void**) &dev_pointCloudBuffer,			sizeof(PointCloud)*width*height);
	hipMalloc((void**) &dev_pointCloudVBO,				sizeof(PointCloud)*width*height);
	hipMalloc((void**) &dev_triangulationIBO,			sizeof(triangleIndecies)*width*height*2);
	hipMalloc((void**) &dev_triangulationIBOCompact,	sizeof(triangleIndecies)*width*height*2);

	cuImageWidth = width;
	cuImageHeight = height;

}

// Free all allocated buffers and close out environment
__host__ void cleanupCuda()
{
	if(imagePBO) deletePBO(&imagePBO);

	hipFree(dev_colorImageBuffer);
	hipFree(dev_depthImageBuffer);
	hipFree(dev_pointCloudBuffer);
	hipFree(dev_pointCloudVBO);
	hipFree(dev_triangulationIBO);
	hipFree(dev_triangulationIBOCompact);
	cuImageWidth = 0;
	cuImageHeight = 0;

	hipDeviceReset();

}

// Copies a depth image to the GPU buffer. 
// Returns false if width and height do not match buffer size set by initCuda(), true if success
__host__ bool pushDepthArrayToBuffer(DPixel* hDepthArray, int width, int height)
{
	if(width != cuImageWidth || height != cuImageHeight)
		return false;//Buffer wrong size

	hipMemcpy(dev_depthImageBuffer, hDepthArray, sizeof(DPixel)*width*height, hipMemcpyHostToDevice);
	return true;
}

// Copies a color image to the GPU buffer. 
// Returns false if width and height do not match buffer size set by initCuda(), true if success
__host__ bool pushColorArrayToBuffer(ColorPixel* hColorArray, int width, int height)
{
	if(width != cuImageWidth || height != cuImageHeight)
		return false; //Buffer wrong size

	hipMemcpy((void*)dev_colorImageBuffer, hColorArray, sizeof(ColorPixel)*width*height, hipMemcpyHostToDevice);
	return true;
}

// Converts the color and depth images currently in GPU buffers into point cloud buffer
__host__ void convertToPointCloud()
{
	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(cuImageWidth)/float(tileSize)), 
		(int)ceil(float(cuImageHeight)/float(tileSize)));

	makePointCloud<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_colorImageBuffer, dev_depthImageBuffer, cuImageWidth, cuImageHeight, dev_pointCloudBuffer);
}

// Computes normals for point cloud in buffer and writes back to the point cloud buffer.
__host__ void computePointCloudNormals()
{
	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(cuImageWidth)/float(tileSize)), 
		(int)ceil(float(cuImageHeight)/float(tileSize)));

	computePointNormals<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_pointCloudBuffer, cuImageWidth, cuImageHeight);
}


// Draws depth image buffer to the texture.
// Texture width and height must match the resolution of the depth image.
// Returns false if width or height does not match, true otherwise
bool drawDepthImageBufferToPBO(float4* dev_PBOpos, int texWidth, int texHeight)
{
	if(texWidth != cuImageWidth || texHeight != cuImageHeight)
		return false;

	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendDepthImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_depthImageBuffer);

	return true;
}

// Draws color image buffer to the texture.
// Texture width and height must match the resolution of the color image.
// Returns false if width or height does not match, true otherwise
// dev_PBOpos must be a CUDA device pointer
bool drawColorImageBufferToPBO(float4* dev_PBOpos, int texWidth, int texHeight)
{
	if(texWidth != cuImageWidth || texHeight != cuImageHeight)
		return false;

	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendColorImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_colorImageBuffer);

	return true;
}


// Renders various debug information about the 2D point cloud buffer to the texture.
// Texture width and height must match the resolution of the point cloud buffer.
// Returns false if width or height does not match, true otherwise
__host__ bool drawPCBToPBO(float4* dptrPosition, float4* dptrColor, float4* dptrNormal, int texWidth, int texHeight)
{
	if(texWidth != cuImageWidth || texHeight != cuImageHeight)
		return false;

	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid( (int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)) );

	sendPCBToPBOs<<<fullBlocksPerGrid, threadsPerBlock>>>(dptrPosition, dptrColor, dptrNormal, glm::vec2(texWidth, texHeight), dev_pointCloudBuffer);

	return true;
}


// Takes a device pointer to the point cloud VBO and copies the contents of the PointCloud buffer to the VBO using stream compaction.
// See: http://nvlabs.github.io/cub/structcub_1_1_device_select.html
__host__ int compactPointCloudToVBO(PointCloud* vbo) {
	int numValid;

	thrust::device_ptr<PointCloud> dp_buffer(dev_pointCloudBuffer);
	thrust::device_ptr<PointCloud> dp_vbo(dev_pointCloudVBO);
	thrust::device_ptr<PointCloud> last = thrust::copy_if(dp_buffer, dp_buffer+(cuImageWidth*cuImageHeight), dp_vbo, IsValidPoint());

	numValid = last - dp_vbo;

	hipMemcpy(vbo, dev_pointCloudVBO, numValid*sizeof(PointCloud), hipMemcpyDeviceToDevice);
	return numValid;
}



int triangulatePCB(triangleIndecies* ibo, float maxTriangleEdgeLength)
{	
	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize, 2);//(x, y) are position of upper left vertex of triangle. (z) is 0 for upper right hand triangle, 1 for lower left hand
	dim3 fullBlocksPerGrid( (int)ceil(float(cuImageWidth)/float(tileSize)), 
		(int)ceil(float(cuImageHeight)/float(tileSize)) );

	triangulationKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_pointCloudBuffer, dev_triangulationIBO, 
		glm::vec2(cuImageWidth, cuImageHeight), maxTriangleEdgeLength);


	thrust::device_ptr<triangleIndecies> dp_buffer(dev_triangulationIBO);
	thrust::device_ptr<triangleIndecies> dp_ibo(dev_triangulationIBOCompact);
	//thrust::device_ptr<triangleIndecies> last = thrust::copy_if(dp_buffer, dp_buffer+(2*cuImageWidth*cuImageHeight), dp_ibo, IsValidTriangle());

	//int numValid = last - dp_ibo;

	hipMemcpy(ibo, dev_triangulationIBO, 2*cuImageHeight*cuImageWidth*sizeof(triangleIndecies), hipMemcpyDeviceToDevice);
	//return numValid;
	return 2*cuImageHeight*cuImageWidth;

}

// Takes a device pointer to the point cloud VBO and copies the contents of the PointCloud buffer to the VBO using stream compaction.
// See: http://nvlabs.github.io/cub/structcub_1_1_device_select.html
__host__ void copyPointCloudToVBO(PointCloud* vbo) {
	hipMemcpy(vbo, dev_pointCloudBuffer, cuImageWidth*cuImageHeight*sizeof(PointCloud), hipMemcpyDeviceToDevice);
}