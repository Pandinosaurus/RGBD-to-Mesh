#include "hip/hip_runtime.h"
#include "preprocessing.h"

__global__ void buildVMapNoFilterKernel(rgbd::framework::DPixel* dev_depthBuffer, VMapSOA vmapSOA, int xRes, int yRes,
										rgbd::framework::Intrinsics intr, float maxDepth) 
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;

	if(u < xRes && v < yRes) 
	{
		int i = (v * xRes) + u;

		float x = HIP_NAN_F;
		float y = HIP_NAN_F;
		float z = dev_depthBuffer[i].depth * 0.001f;

		if (z > 0.001f && z < maxDepth) {//Exclude zero or negative depths.  

			x = (u - intr.cx) * z / intr.fx;
			y = (v - intr.cy) * z / intr.fy;
		} else{
			z = HIP_NAN_F;
		}

		//Write to SOA in memory coallesed way
		vmapSOA.x[0][i] = x;
		vmapSOA.y[0][i] = y;
		vmapSOA.z[0][i] = z;

	}
}

__host__ void buildVMapNoFilterCUDA(rgbd::framework::DPixel* dev_depthBuffer, VMapSOA vmapSOA, int xRes, int yRes, 
									rgbd::framework::Intrinsics intr, float maxDepth)
{

	int tileSize = 16;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(xRes)/float(tileSize)), 
		(int)ceil(float(yRes)/float(tileSize)));

	buildVMapNoFilterKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_depthBuffer, vmapSOA, xRes, yRes, intr, maxDepth);

}

struct KernelWindow
{
	float kernel[MAX_FILTER_WINDOW_SIZE];
};

__global__ void gaussianKernel1(rgbd::framework::DPixel* dev_depthBuffer, VMapSOA vmapSOA, int xRes, int yRes,
								float maxDepth, float sigma, int window, KernelWindow precomputed)
{
	extern __shared__ float sharedRow[];

	int row = blockIdx.x;
	int col = threadIdx.x;
	int i = row*xRes + col;

	if(col < xRes)
	{
		sharedRow[col] = dev_depthBuffer[i].depth*0.001f;
	}

	__syncthreads();

	//Row loaded in shared memory.

	float weightSum = 0.0f;
	float accum = 0.0f;
	int leftEdge = MAX(0,(col - (window>>1)));
	int rightEdge = MIN((xRes-1), (col + (window>>1)));
	for(int x = leftEdge, int i = 0; x <= rightEdge; ++x, ++i)
	{
		accum += sharedRow[x]*precomputed.kernel[i];
		weightSum += precomputed.kernel[i];
	}

	if(weightSum > 0.0){
		accum /= weightSum;//Normalize
		vmapSOA.z[0][i] = accum;
	}else{
		vmapSOA.z[0][i] = HIP_NAN_F;
	}
}



__host__ void buildVMapGaussianFilterCUDA(rgbd::framework::DPixel* dev_depthBuffer, VMapSOA vmapSOA, int xRes, int yRes, 
										  rgbd::framework::Intrinsics intr, float maxDepth, float sigma, int window)
{
	//Seperable kernel. Rows first. For simplicity and efficiency, assume sensor resolution less than 1080p. 
	//A reasonable assumption for current state of technology
	if(xRes < 1024 && yRes < 1024)
	{
		if(window > MAX_FILTER_WINDOW_SIZE)
		{
			throw new std::exception("Error: Filter window too big");
		}

		dim3 threadsPerBlock(xRes);
		dim3 fullBlocksPerGrid(yRes);
		int sharedMemSize = sizeof(float)*xRes;

		KernelWindow precomputedWindow;
		int center = window >> 1;
		for(int i = 0; i < window; i++)
		{
			float dist = center - i;

			precomputedWindow.kernel[i] = expf(-dist*dist/(2*sigma));
		}

		gaussianKernel1<<<fullBlocksPerGrid, threadsPerBlock, sharedMemSize>>>(dev_depthBuffer, vmapSOA, xRes, yRes, maxDepth, sigma, window, precomputedWindow);

	}else{
		throw new std::exception("Error: Input image exceeds maximum dimension.");
	}
}



__global__ void rgbAOSToSOAKernel(rgbd::framework::ColorPixel* dev_colorPixels, 
								  RGBMapSOA rgbSOA, int xRes, int yRes)
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;

	if(u < xRes && v < yRes) 
	{
		int i = (v * xRes) + u;

		rgbd::framework::ColorPixel color = dev_colorPixels[i];
		rgbSOA.r[i] = color.r / 255.0f;
		rgbSOA.g[i] = color.g / 255.0f;
		rgbSOA.b[i] = color.b / 255.0f;
	}

}

__host__ void rgbAOSToSOACUDA(rgbd::framework::ColorPixel* dev_colorPixels, 
							  RGBMapSOA rgbSOA, int xRes, int yRes)
{
	int tileSize = 16;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(xRes)/float(tileSize)), 
		(int)ceil(float(yRes)/float(tileSize)));

	rgbAOSToSOAKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_colorPixels, rgbSOA, xRes, yRes);
}


//Subsamples float3 SOA by 1/2 and stores in dest
//Threads are parallel by `
__global__ void subsampleVMAPKernel(float* x_src, float* y_src, float* z_src, 
									float* x_dest, float* y_dest, float* z_dest,
									int xRes_src, int yRes_src)
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;
	int xRes_dest = xRes_src >> 1;
	int yRes_dest = yRes_src >> 1;

	if(u < xRes_dest  && v < yRes_dest) 
	{
		int i_src = (v<<1)*xRes_src+(u<<1);
		int i_dest = (v * xRes_dest) + u;

		x_dest[i_dest] = x_src[i_src];
		y_dest[i_dest] = y_src[i_src];
		z_dest[i_dest] = z_src[i_src];


	}
}

__host__ void buildVMapPyramidCUDA(VMapSOA dev_vmapSOA, int xRes, int yRes, int numLevels)
{
	int tileSize = 16;

	for(int i = 0; i < numLevels - 1; ++i)
	{
		dim3 threadsPerBlock(tileSize, tileSize);
		dim3 fullBlocksPerGrid((int)ceil(float(xRes>>(1+i))/float(tileSize)), 
			(int)ceil(float(yRes>>(1+i))/float(tileSize)));


		subsampleVMAPKernel<<<fullBlocksPerGrid,threadsPerBlock>>>(dev_vmapSOA.x[i], dev_vmapSOA.y[i], dev_vmapSOA.z[i],
			dev_vmapSOA.x[i+1], dev_vmapSOA.y[i+1], dev_vmapSOA.z[i+1],
			xRes>>i, yRes>>i);
	}

}