#include "hip/hip_runtime.h"
#include "normal_estimates.h"

#pragma region Simple Normals Calculation

__global__ void simpleNormalsKernel(float* x_vert, float* y_vert, float* z_vert, 
									float* x_norm, float* y_norm, float* z_norm,
									float* curvature,
									int xRes, int yRes)
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;

	int i = v * xRes + u;

	if(u < xRes && v < yRes){

		glm::vec3 norm = glm::vec3(HIP_NAN_F);

		if(u < xRes - 1 && v < yRes - 1 && u > 0 && v > 0)
		{

			//Diff to right
			float dx1 = x_vert[i+1] - x_vert[i-1];
			float dy1 = y_vert[i+1] - y_vert[i-1];
			float dz1 = z_vert[i+1] - z_vert[i-1];

			//Diff to bottom
			float dx2 = x_vert[i+xRes] - x_vert[i-xRes];
			float dy2 = y_vert[i+xRes] - y_vert[i-xRes];
			float dz2 = z_vert[i+xRes] - z_vert[i-xRes];

			//d1 cross d2
			norm.x = dy1*dz2-dz1*dy2;
			norm.y = dz1*dx2-dx1*dz2;
			norm.z = dx1*dy2-dy1*dx2;

			if(norm.z > 0.0f)
			{
				//Flip towards camera
				norm = -norm;
			}

			norm = glm::normalize(norm);

		}

		x_norm[i] = norm.x;
		y_norm[i] = norm.y;
		z_norm[i] = norm.z;
		curvature[i] = 0.0f;//filler. Simple normals has no means of estimating curvature


	}

}

__host__ void simpleNormals(Float3SOAPyramid vmap, Float3SOAPyramid nmap, Float1SOAPyramid curvaturemap, int numLevels, int xRes, int yRes)
{
	int tileSize = 16;

	for(int i = 0; i < numLevels; ++i)
	{
		dim3 threadsPerBlock(tileSize, tileSize);
		dim3 fullBlocksPerGrid((int)ceil(float(xRes>>i)/float(tileSize)), 
			(int)ceil(float(yRes>>i)/float(tileSize)));


		simpleNormalsKernel<<<fullBlocksPerGrid,threadsPerBlock>>>(vmap.x[i], vmap.y[i], vmap.z[i],
			nmap.x[i], nmap.y[i], nmap.z[i], curvaturemap.x[i],
			xRes>>i, yRes>>i);
	}
}

#pragma endregion

#pragma region  Filtered Average Gradient


__global__ void normalsFromGradientKernel(float* horizontalGradientX, float* horizontalGradientY, float* horizontalGradientZ,
										  float* vertGradientX, float* vertGradientY, float* vertGradientZ,
										  float* x_norm, float* y_norm, float* z_norm, float* curvature,
										  int xRes, int yRes)
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;

	int i = v * xRes + u;

	if(u < xRes && v < yRes){

		glm::vec3 norm = glm::normalize(glm::cross(glm::vec3(vertGradientX[i], vertGradientY[i], vertGradientZ[i]), 
			glm::vec3(horizontalGradientX[i], horizontalGradientY[i], horizontalGradientZ[i])));

		if(norm.z > 0.0f)
		{
			//Flip towards camera
			norm = -norm;
		}

		x_norm[i] = norm.x;
		y_norm[i] = norm.y;
		z_norm[i] = norm.z;
		curvature[i] = 0.0f;//filler. Simple normals has no means of estimating curvature


	}
}

__host__ void computeAverageGradientNormals(Float3SOAPyramid horizontalGradient, Float3SOAPyramid vertGradient, 
											Float3SOAPyramid nmap, Float1SOAPyramid curvature, int xRes, int yRes)
{
	int tileSize = 16;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(xRes)/float(tileSize)), 
		(int)ceil(float(yRes)/float(tileSize)));


	normalsFromGradientKernel<<<fullBlocksPerGrid,threadsPerBlock>>>(horizontalGradient.x[0], horizontalGradient.y[0], horizontalGradient.z[0],
		vertGradient.x[0], vertGradient.y[0], vertGradient.z[0],
		nmap.x[0], nmap.y[0], nmap.z[0], curvature.x[0],
		xRes, yRes);

}

#pragma endregion

#pragma region Eigen Normal Calculation


#pragma endregion