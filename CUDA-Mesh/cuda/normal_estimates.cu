#include "hip/hip_runtime.h"
#include "normal_estimates.h"

#pragma region Simple Normals Calculation

__global__ void simpleNormalsKernel(float* x_vert, float* y_vert, float* z_vert, 
									float* x_norm, float* y_norm, float* z_norm,
									float* curvature,
									int xRes, int yRes)
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;

	int i = v * xRes + u;

	if(u < xRes && v < yRes){

		glm::vec3 norm = glm::vec3(HIP_NAN_F);

		if(u < xRes - 1 && v < yRes - 1 && u > 0 && v > 0)
		{

			//Diff to right
			float dx1 = x_vert[i+1] - x_vert[i-1];
			float dy1 = y_vert[i+1] - y_vert[i-1];
			float dz1 = z_vert[i+1] - z_vert[i-1];

			//Diff to bottom
			float dx2 = x_vert[i+xRes] - x_vert[i-xRes];
			float dy2 = y_vert[i+xRes] - y_vert[i-xRes];
			float dz2 = z_vert[i+xRes] - z_vert[i-xRes];

			//d1 cross d2
			norm.x = dy1*dz2-dz1*dy2;
			norm.y = dz1*dx2-dx1*dz2;
			norm.z = dx1*dy2-dy1*dx2;

			if(norm.z > 0.0f)
			{
				//Flip towards camera
				norm = -norm;
			}

			norm = glm::normalize(norm);

		}

		x_norm[i] = norm.x;
		y_norm[i] = norm.y;
		z_norm[i] = norm.z;
		curvature[i] = 0.0f;//filler. Simple normals has no means of estimating curvature


	}

}

__host__ void simpleNormals(Float3SOAPyramid vmap, Float3SOAPyramid nmap, Float1SOAPyramid curvaturemap, int numLevels, int xRes, int yRes)
{
	int tileSize = 16;

	for(int i = 0; i < numLevels; ++i)
	{
		dim3 threadsPerBlock(tileSize, tileSize);
		dim3 fullBlocksPerGrid((int)ceil(float(xRes>>i)/float(tileSize)), 
			(int)ceil(float(yRes>>i)/float(tileSize)));


		simpleNormalsKernel<<<fullBlocksPerGrid,threadsPerBlock>>>(vmap.x[i], vmap.y[i], vmap.z[i],
			nmap.x[i], nmap.y[i], nmap.z[i], curvaturemap.x[i],
			xRes>>i, yRes>>i);
	}
}

#pragma endregion

#pragma region  Filtered Average Gradient


__global__ void normalsFromGradientKernel(float* horizontalGradientX, float* horizontalGradientY, float* horizontalGradientZ,
										  float* vertGradientX, float* vertGradientY, float* vertGradientZ,
										  float* x_norm, float* y_norm, float* z_norm, float* curvature,
										  int xRes, int yRes)
{
	int u = (blockIdx.x * blockDim.x) + threadIdx.x;
	int v = (blockIdx.y * blockDim.y) + threadIdx.y;

	int i = v * xRes + u;

	if(u < xRes && v < yRes){

		glm::vec3 norm = glm::normalize(glm::cross(glm::vec3(vertGradientX[i], vertGradientY[i], vertGradientZ[i]), 
			glm::vec3(horizontalGradientX[i], horizontalGradientY[i], horizontalGradientZ[i])));

		if(norm.z > 0.0f)
		{
			//Flip towards camera
			norm = -norm;
		}

		x_norm[i] = norm.x;
		y_norm[i] = norm.y;
		z_norm[i] = norm.z;
		curvature[i] = 0.0f;//filler. Simple normals has no means of estimating curvature


	}
}

__host__ void computeAverageGradientNormals(Float3SOAPyramid horizontalGradient, Float3SOAPyramid vertGradient, 
											Float3SOAPyramid nmap, Float1SOAPyramid curvature, int xRes, int yRes)
{
	int tileSize = 16;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(xRes)/float(tileSize)), 
		(int)ceil(float(yRes)/float(tileSize)));


	normalsFromGradientKernel<<<fullBlocksPerGrid,threadsPerBlock>>>(horizontalGradient.x[0], horizontalGradient.y[0], horizontalGradient.z[0],
		vertGradient.x[0], vertGradient.y[0], vertGradient.z[0],
		nmap.x[0], nmap.y[0], nmap.z[0], curvature.x[0],
		xRes, yRes);

}

#pragma endregion

#pragma region Eigen Normal Calculation

#define PCA_TILE_SIZE 16
#define PCA_WINDOW_RADIUS 2
#define PCA_MIN_NEIGHBORS 16

__device__ float distanceSq(glm::vec3 p1, glm::vec3 p2)
{
	float dx = p1.x-p2.x;
	float dy = p1.y-p2.y;
	float dz = p1.z-p2.z;

	return dx*dx+dy*dy+dz*dz;
}

__global__ void pcaNormalsKernel(float* vmapX, float* vmapY, float* vmapZ, float* nmapX, float* nmapY, float* nmapZ, float* curvature,
								 int xRes, int yRes, float radiusMetersSq)
{
	__shared__ float s_positions[3][PCA_TILE_SIZE+2*PCA_WINDOW_RADIUS][PCA_TILE_SIZE+2*PCA_WINDOW_RADIUS];


	//Upper left corner of aligned loading block
	int loadBx = blockIdx.x*blockDim.x-PCA_TILE_SIZE/2;
	int loadBy = blockIdx.y*blockDim.y-PCA_TILE_SIZE/2;

	//Index of this thread's work target
	int resultsX = blockIdx.x*blockDim.x + threadIdx.x;
	int resultsY = blockIdx.y*blockDim.y + threadIdx.y;
	int i = resultsX + xRes*resultsY;

	int loadX = threadIdx.x + 16 * (threadIdx.y % 2);
	//Offset to shared memory is threadIdx.x-PCA_WINDOW_RADIUS, threadIdx.y-PCA_WINDOW_RADIUS
	if(loadX >= PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS && loadX < PCA_TILE_SIZE*3/2 + PCA_WINDOW_RADIUS){
		//Is in horizontal range. Only need to perform vertical check in loop

#pragma unroll
		for(int istep = 0; istep < 3; ++istep)
		{
			int loadY = istep*PCA_TILE_SIZE/2 + threadIdx.y/2;
			if(loadY >= PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS && loadY < PCA_TILE_SIZE*3/2 + PCA_WINDOW_RADIUS)
			{
				bool loadIdInRange = (loadBy + loadY) > 0 && (loadBy + loadY) < yRes && (loadBx + loadX) > 0 && (loadBx + loadX) < xRes;

				int loadI = (loadBy + loadY)*xRes + (loadBx + loadX);

				//Tiles garunteed to be in range of original image by block layout
				s_positions[0][loadY - (PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS)][loadX - (PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS)] = loadIdInRange?vmapX[loadI]:HIP_NAN_F;
				s_positions[1][loadY - (PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS)][loadX - (PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS)] = loadIdInRange?vmapY[loadI]:HIP_NAN_F;
				s_positions[2][loadY - (PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS)][loadX - (PCA_TILE_SIZE/2 - PCA_WINDOW_RADIUS)] = loadIdInRange?vmapZ[loadI]:HIP_NAN_F;
			}
		}
	}

	__syncthreads();

	//Load done
	//Compute centroid. 
	//Use center point to determine point range
	int sx = threadIdx.x+PCA_WINDOW_RADIUS;
	int sy = threadIdx.y+PCA_WINDOW_RADIUS;
	glm::vec3 centerPos = glm::vec3(s_positions[0][sy][sx],
		s_positions[1][sy][sx],
		s_positions[2][sy][sx]);

	int neighborCount = 0;
	glm::vec3 centroid = glm::vec3();
#pragma unroll
	for(int x = -PCA_WINDOW_RADIUS; x <= PCA_WINDOW_RADIUS; ++x)
	{
#pragma unroll
		for(int y = -PCA_WINDOW_RADIUS; y <= PCA_WINDOW_RADIUS; ++y)
		{
			glm::vec3 p = glm::vec3(s_positions[0][sy+y][sx+x],
				s_positions[1][sy+y][sx+x],
				s_positions[2][sy+y][sx+x]);
			glm::vec3 diff = p-centerPos;
			if(glm::dot(diff,diff) <= radiusMetersSq)
			{
				//In range
				neighborCount++;
				centroid += p;
			}
		}
	}
	centroid /= neighborCount;

	//At this point, we have a true centroid


	nmapX[i] = centroid.x;
	nmapY[i] = centroid.y;//
	nmapZ[i] = neighborCount/25.0;//
	curvature[i] = 0.0f;

}


__host__ void computePCANormals(Float3SOAPyramid vmap, Float3SOAPyramid nmap, Float1SOAPyramid curvaturemap, 
								int xRes, int yRes, float radiusMeters)
{
	assert(PCA_WINDOW_RADIUS < PCA_TILE_SIZE / 2);

	dim3 threads(PCA_TILE_SIZE, PCA_TILE_SIZE);
	dim3 blocks((int)ceil(float(xRes)/float(PCA_TILE_SIZE)), 
		(int)ceil(float(yRes)/float(PCA_TILE_SIZE)));


	pcaNormalsKernel<<<blocks,threads>>>(vmap.x[0], vmap.y[0], vmap.z[0], nmap.x[0], nmap.y[0], nmap.z[0], curvaturemap.x[0],
		xRes, yRes, radiusMeters*radiusMeters);


}

#pragma endregion