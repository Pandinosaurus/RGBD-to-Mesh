#include "hip/hip_runtime.h"
#include "plane_segmentation.h"


__device__ glm::vec3 normalFrom3x3Covar(glm::mat3 A, float& curvature) {
	// Given a (real, symmetric) 3x3 covariance matrix A, returns the eigenvector corresponding to the min eigenvalue
	// (see: http://en.wikipedia.org/wiki/Eigenvalue_algorithm#3.C3.973_matrices)
	glm::vec3 eigs;
	glm::vec3 normal = glm::vec3(0.0f);

	float p1 = A[0][1]*A[0][1] + A[0][2]*A[0][2] + A[1][2]*A[1][2];
	if (abs(p1) < 0.00001f) { // A is diagonal
		eigs = glm::vec3(A[0][0], A[1][1], A[2][2]);

		float tmp;
		int i, eig_i;
		// sorting: swap first pair if necessary, then second pair, then first pair again
		for (i=0; i<3; i++) {
			eig_i = i%2;
			tmp = eigs[eig_i];
			eigs[eig_i] = glm::max(tmp, eigs[eig_i+1]);
			eigs[eig_i+1] = glm::min(tmp, eigs[eig_i+1]);
		}
	} else {
		float q = (A[0][0] + A[1][1] + A[2][2])/3.0f; // mean(trace(A))
		float p2 = (A[0][0]-q)*(A[0][0]-q) + (A[1][1]-q)*(A[1][1]-q) + (A[2][2]-q)*(A[2][2]-q)+ 2*p1;
		float p = sqrt(p2/6);
		glm::mat3 B = (1/p) * (A-q*glm::mat3(1.0f));
		float r = glm::determinant(B)/2;
		// theoretically -1 <= r <= 1, but clamp in case of numeric error
		float phi;
		if (r <= -1) {
			phi = PI_F / 3;
		} else if (r >= 1) { 
			phi = 0;
		} else {
			phi = glm::acos(r)/3;
		}
		eigs[0] = q + 2*p*glm::cos(phi);
		eigs[2] = q + 2*p*glm::cos(phi + 2*PI_F/3);
		eigs[1] = 3*q - eigs[0] - eigs[2];

	}



	//N = (A-eye(3)*eig1)*(A(:,1)-[1;0;0]*eig2);
	glm::mat3 Aeig1 = A;
	Aeig1[0][0] -= eigs[0];
	Aeig1[1][1] -= eigs[0];
	Aeig1[2][2] -= eigs[0];
	normal = Aeig1*(A[0] - glm::vec3(eigs[1],0.0f,0.0f));

	// check if point cloud region is "flat" enough
	curvature = eigs[2]/(eigs[0]+eigs[1]+eigs[2]);


	float length = glm::length(normal);
	normal /= length;
	return normal;
}


#pragma region Histogram Two-D

__global__ void normalHistogramKernel(float* normX, float* normY, int* histogram, int xRes, int yRes, int xBins, int yBins)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	if( i  < xRes*yRes)
	{
		float x = normX[i];
		float y = normY[i];
		if(x == x && y == y)//Will be false if NaN
		{
			//int xI = (x+1.0f)*0.5f*xBins;//x in range of -1 to 1. Map to 0 to 1.0 and multiply by number of bins
			//int yI = (y+1.0f)*0.5f*yBins;//x in range of -1 to 1. Map to 0 to 1.0 and multiply by number of bins
			//int xI = acos(x)*PI_INV_F*xBins;
			//int yI = acos(y)*PI_INV_F*yBins;
			float azimuth = acosf(x/sqrtf(1.0f-y*y));
			int xI = azimuth*PI_INV_F*xBins;
			int yI = acos(y)*PI_INV_F*yBins;

			atomicAdd(&histogram[yI*xBins + xI], 1);
		}
	}
}



__host__ void computeNormalHistogram(float* normX, float* normY, int* histogram, int xRes, int yRes, int xBins, int yBins)
{
	int blockLength = 256;

	dim3 threads(blockLength);
	dim3 blocks((int)(ceil(float(xRes*yRes)/float(blockLength))));


	normalHistogramKernel<<<blocks,threads>>>(normX, normY, histogram, xRes, yRes, xBins, yBins);

}

__global__ void clearHistogramKernel(int* histogram, int length)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if(i < length)
	{
		histogram[i] = 0;
	}
}

__host__ void clearHistogram(int* histogram, int xBins, int yBins)
{
	int blockLength = 256;

	dim3 threads(blockLength);
	dim3 blocks((int)(ceil(float(xBins*yBins)/float(blockLength))));

	clearHistogramKernel<<<blocks,threads>>>(histogram, xBins*yBins);

}

#pragma endregion


#pragma region ACos Histogram One-D

//TODO: Shared memory
__global__ void ACosHistogramKernel(float* cosineValue, int* histogram, int valueCount, int numBins)
{
	extern __shared__ int s_hist[];
	s_hist[threadIdx.x] = 0;
	__syncthreads();

	int valueI = threadIdx.x + blockDim.x * blockIdx.x;

	if(valueI < valueCount)
	{
		float angle = acosf(cosineValue[valueI]);

		if(angle == angle){
			int histIndex = angle*PI_INV_F*numBins;
			if(histIndex >= 0 && histIndex < numBins)//Sanity check
				atomicAdd(&s_hist[histIndex], 1);
		}
	}

	__syncthreads();

	atomicAdd(&histogram[threadIdx.x], s_hist[threadIdx.x]);
}

__host__ void ACosHistogram(float* cosineValue, int* histogram, int valueCount, int numBins)
{
	int blockLength = numBins;

	dim3 threads(blockLength);
	dim3 blocks((int)(ceil(float(valueCount)/float(blockLength))));

	ACosHistogramKernel<<<blocks,threads, numBins*sizeof(int)>>>(cosineValue, histogram, valueCount, numBins);
}

#pragma endregion


#pragma region Simple Histogram Peak Detection

__global__ void gaussianSubtractionPeakDetectionKernel(int* histx, int* histy, int* histz, int* peaksX, int* peaksY, int* peaksZ, 
													   int histLength, int maxPeaks, int minPeakCount, glm::vec3 sigma2inv)
{
	//Setup shared buffers
	extern __shared__ int s_temp[];
	int* s_hist = s_temp;
	int* s_max = s_hist + histLength;
	int* s_maxI = s_max + histLength/2;
	int* s_peaks = s_maxI + histLength/2;

	//Load histogram from different location for each block
	float sig = sigma2inv[blockIdx.x];
	if(blockIdx.x == 0)
		s_hist[threadIdx.x] = histx[threadIdx.x];
	else if(blockIdx.x == 1)
		s_hist[threadIdx.x] = histy[threadIdx.x];
	else //if(blockIdx.x == 2)
		s_hist[threadIdx.x] = histz[threadIdx.x];

	//clear peaks
	if(threadIdx.x < maxPeaks)
		s_peaks[threadIdx.x] = -1;

	__syncthreads();
	//====Load/Init Complete=====
	//====Begin Peak Loop =======

	//For up to the maximum number of peaks
	for(int peaki = 0; peaki < maxPeaks; ++peaki)
	{

#pragma region Maximum Finder
		//========Compute maximum=======
		//First step loads from main hist, so do outside loop
		int halfpoint = histLength >> 1;
		int thread2 = threadIdx.x + halfpoint;

		if(threadIdx.x < halfpoint)
		{
			int temp = s_hist[thread2];
			bool leftSmaller = (s_hist[threadIdx.x] < temp);
			s_max[threadIdx.x] = leftSmaller?temp:s_hist[threadIdx.x];
			s_maxI[threadIdx.x] = leftSmaller?thread2:threadIdx.x;
		}
		__syncthreads();
		while(halfpoint > 0)
		{
			halfpoint >>= 1;
			if(threadIdx.x < halfpoint)
			{
				thread2 = threadIdx.x + halfpoint;
				int temp = s_max[thread2];
				if (temp > s_max[threadIdx.x]) {
					s_max[threadIdx.x] = temp;
					s_maxI[threadIdx.x] = s_maxI[thread2];
				}
			}
			__syncthreads();
		}

		//========Compute maximum End=======
#pragma endregion

		if(threadIdx.x == 0)
		{
			s_peaks[peaki] = s_maxI[0];
		}

		if(s_max[0] < minPeakCount)
			break;//done. No more peaks to find

		//=====Subtract gaussian model=====
		int diff = (threadIdx.x-s_maxI[peaki]);
		s_hist[threadIdx.x] -= s_max[0] * expf(-diff*diff*sig);

		__syncthreads();
	}

	//Writeback
	if(threadIdx.x < maxPeaks)
	{
		if(blockIdx.x == 0)
			peaksX[threadIdx.x] = s_peaks[threadIdx.x];
		else if(blockIdx.x == 1)
			peaksY[threadIdx.x] = s_peaks[threadIdx.x];
		else //if(blockIdx.x == 2)
			peaksZ[threadIdx.x] = s_peaks[threadIdx.x];
	}
}

__host__ void gaussianSubtractionPeakDetection(Int3SOA decoupledHist, Int3SOA peakIndex, int histSize, int maxPeaks, int minPeakCount, glm::vec3 sigmas)
{
	assert(histSize > 32);
	assert(!(histSize & (histSize - 1))); //Assert is power of two
	assert(histSize % 32 == 0);//Assert is multiple of 32

	int sharedSize = (histSize*2 + maxPeaks)*sizeof(int);
	dim3 threads(histSize);
	dim3 blocks(3);

	gaussianSubtractionPeakDetectionKernel<<<blocks,threads,sharedSize>>>(decoupledHist.x, decoupledHist.y, decoupledHist.z, 
		peakIndex.x, peakIndex.y, peakIndex.z, histSize, maxPeaks, minPeakCount, 1.0f/(2.0f*sigmas*sigmas));

}


#pragma endregion

#pragma region Histogram Peak Detection Two-D

__global__ void normalHistogramPrimaryPeakDetectionKernel(int* histogram, int xBins, int yBins, Float3SOA peaks, int maxPeaks, 
														  int exclusionRadius, int minPeakHeight)
{	
	extern __shared__ int s_temp[];
	int* s_hist = s_temp;
	int* s_max = s_hist + xBins*yBins;
	int* s_maxI = s_max + (xBins*yBins)/2;

	int index = threadIdx.x + threadIdx.y*xBins;
	//Load histogram
	s_hist[index] = histogram[index];
	__syncthreads();


	//Find local maxima
	bool localMax = false;

	if(s_hist[index] > minPeakHeight)
	{
		localMax = true;
		if(threadIdx.x > 0)
			if(s_hist[index - 1] > s_hist[index])
				localMax = false;


		if(threadIdx.x < xBins-1)
			if(s_hist[index + 1] > s_hist[index])
				localMax = false;

		if(threadIdx.y > 0)
			if(s_hist[index - xBins] > s_hist[index])
				localMax = false;

		if(threadIdx.y > yBins-1)
			if(s_hist[index + xBins] > s_hist[index])
				localMax = false;
	}

	float totalCount = 0.0f;
	float xPos = 0.0f;
	float yPos = 0.0f;
	if(localMax)
	{

		for(int x = -1; x <= 1; ++x)
		{
			int tx = threadIdx.x + x;
			for(int y = -1; y <= 1; ++y)
			{
				int ty = threadIdx.y + y;
				if(tx >= 0 && tx < xBins && ty >= 0 && ty < yBins)
				{
					int binCount = s_hist[tx + ty*xBins];
					totalCount += binCount;
					xPos += binCount*tx;
					yPos += binCount*ty;

				}
			}

		}
		xPos /= totalCount;
		yPos /= totalCount;

	}

	__syncthreads();

	if(!localMax)
	{
		s_hist[index] = 0;//clear all non-local max histograms

		//DEBUG
		//histogram[index] = 0;
	}
	__syncthreads();
	//Preprocessing complete

	//=========Peak detection Loop===========
	int histLength = xBins*yBins;
	for(int peakNum = 0; peakNum < maxPeaks; ++peakNum)
	{

#pragma region Maximum Finder
		//========Compute maximum=======
		//First step loads from main hist, so do outside loop
		int halfpoint = histLength >> 1;
		int thread2 = index + halfpoint;
		if(index < halfpoint)
		{
			int temp = s_hist[thread2];
			bool leftSmaller = (s_hist[index] < temp);
			s_max[index] = leftSmaller?temp:s_hist[index];
			s_maxI[index] = leftSmaller?thread2:index;
		}
		__syncthreads();
		while(halfpoint > 0)
		{
			halfpoint >>= 1;
			if(index < halfpoint)
			{
				thread2 = index + halfpoint;
				int temp = s_max[thread2];
				if (temp > s_max[index]) {
					s_max[index] = temp;
					s_maxI[index] = s_maxI[thread2];
				}
			}
			__syncthreads();
		}

		//========Compute maximum End=======
#pragma endregion



		//s_maxI[0] now holds the maximum index

		if(s_max[0] < minPeakHeight)
		{
			//Fill remaining slots with -1
			if(index >= peakNum && index < maxPeaks)
			{
				peaks.x[index] = -1;
				peaks.y[index] = -1;
				peaks.z[index] = -1;
			}
			break;
		}

		if(s_maxI[0] == index)
		{
			peaks.x[peakNum] = xPos;
			peaks.y[peakNum] = yPos;
			peaks.z[peakNum] = s_hist[index];
			//DEBUG
			histogram[index] = -(peakNum+1);
		}

		//Distance to max
		int dx = (s_maxI[0] % xBins) - threadIdx.x;
		int dy = (s_maxI[0] / yBins) - threadIdx.y;

		if(dx*dx+dy*dy < exclusionRadius*exclusionRadius)
		{
			s_hist[index] = 0;
		}


		__syncthreads();
	}
}


__host__ void normalHistogramPrimaryPeakDetection(int* histogram, int xBins, int yBins, Float3SOA peaks, int maxPeaks, 
												  int exclusionRadius, int minPeakHeight)
{
	assert(xBins*yBins <= 1024);//For now enforce strict limit. Might be expandable in future, but most efficient like this
	assert(!(xBins*yBins  & (xBins*yBins  - 1))); //Assert is power of two



	dim3 threads(xBins, yBins);
	dim3 blocks(1);

	int sharedMem = xBins*yBins*2*sizeof(int);

	normalHistogramPrimaryPeakDetectionKernel<<<blocks,threads,sharedMem>>>(histogram, xBins, yBins, peaks, 
		maxPeaks, exclusionRadius, minPeakHeight);
}

#pragma endregion

#pragma region Segmentation Two-D

__global__ void segmentNormals2DKernel(Float3SOA rawNormals, Float3SOA rawPositions, 
									   int* normalSegments, float* projectedDistance,
									   int imageWidth, int imageHeight, 
									   int* histogram, int xBins, int yBins, 
									   Float3SOA peaks, int maxPeaks, float maxAngleRange)
{
	extern __shared__ float s_mem[];
	float* s_peaksX = s_mem;
	float* s_peaksY = s_peaksX + maxPeaks;
	float* s_peaksZ = s_peaksY + maxPeaks;

	int index = threadIdx.x + blockIdx.x*blockDim.x;

	if(threadIdx.x < maxPeaks)
	{
		float xi = peaks.x[threadIdx.x];
		float yi = peaks.y[threadIdx.x];
		float x = 0.0f;
		float y = 0.0f;
		float z = 0.0f;

		if(xi >= 0.0f && yi >= 0.0f){

			y = cosf(PI_F*yi/float(yBins));
			x = cosf(PI_F*xi/float(xBins)) * sqrtf(1.0f-y*y);
			z = sqrtf(1.0f-x*x-y*y);
		}

		s_peaksX[threadIdx.x] = x;
		s_peaksY[threadIdx.x] = y;
		s_peaksZ[threadIdx.x] = z;
	}

	__syncthreads();


	if(index < imageWidth*imageHeight)
	{

		glm::vec3 normal = glm::vec3(rawNormals.x[index], rawNormals.y[index], rawNormals.z[index]);
		int bestPeak = -1;
		if(normal.x == normal.x && normal.y == normal.y && normal.z == normal.z)
		{
			//normal is valid
			for(int peakNum = 0; peakNum < maxPeaks; ++peakNum)
			{
				float dotprod = normal.x*s_peaksX[peakNum] + normal.y*s_peaksY[peakNum] + normal.z*s_peaksZ[peakNum];
				float angle = acosf(dotprod);

				if(angle < maxAngleRange)
				{
					bestPeak = peakNum;
					break;
				}
			}
		}

		float projectedD = HIP_NAN_F;//Initialize to NAN
		if(bestPeak >= 0)
		{
			//Peak found, compute projection
			projectedD = s_peaksX[bestPeak]*rawPositions.x[index] 
			+ s_peaksY[bestPeak]*rawPositions.y[index] 
			+ s_peaksZ[bestPeak]*rawPositions.z[index];

		}


		//Writeback
		normalSegments[index] = bestPeak;
		projectedDistance[index] = projectedD;
	}

}

__host__ void segmentNormals2D(Float3SOA rawNormals, Float3SOA rawPositions, 
							   int* normalSegments, float* projectedDistance,int imageWidth, int imageHeight,
							   int* normalHistogram, int xBins, int yBins, 
							   Float3SOA peaks, int maxPeaks, float maxAngleRange)
{
	int blockLength = 512;
	assert(blockLength > maxPeaks);

	dim3 blocks((int) ceil(float(imageWidth*imageHeight)/float(blockLength)));
	dim3 threads(blockLength);

	int sharedCount = sizeof(float)*(3 * maxPeaks);

	segmentNormals2DKernel<<<blocks, threads, sharedCount>>>(rawNormals, rawPositions, normalSegments, projectedDistance, 
		imageWidth, imageHeight, normalHistogram, xBins, yBins, peaks, maxPeaks, maxAngleRange);
}


#pragma endregion

#pragma region Distance Histograms

__global__ void distanceHistogramKernel(int* dev_normalSegments, float* dev_planeProjectedDistanceMap, int xRes, int yRes,
										 int* dev_distanceHistograms, int numMaxNormalSegments, 
										 int histcount, float histMinDist, float histMaxDist)
{
	extern __shared__ int s_temp[];
	int* s_hist = s_temp;

	int index = threadIdx.x + blockIdx.x*blockDim.x;

	int segment = dev_normalSegments[index];
	float dist = dev_planeProjectedDistanceMap[index];
	int histI = -1;
	if(segment >= 0)
	{
		if(dist < histMaxDist && dist >= histMinDist) 
			histI = (dist - histMinDist)*histcount/(histMaxDist-histMinDist);
	}

	//Each thread has locally stored values.
	for(int peak = 0; peak < numMaxNormalSegments; ++peak)
	{
		//reset histogram
		s_temp[threadIdx.x] = 0;
		__syncthreads();

		if(segment == peak && histI >= 0)
		{
			atomicAdd(&s_hist[histI], 1);
		}
	
		__syncthreads();

		atomicAdd(&(dev_distanceHistograms[peak*histcount + threadIdx.x]), s_hist[threadIdx.x]);
	}
}

__host__ void generateDistanceHistograms(int* dev_normalSegments, float* dev_planeProjectedDistanceMap, int xRes, int yRes,
										 int** dev_distanceHistograms, int numMaxNormalSegments, 
										 int histcount, float histMinDist, float histMaxDist)
{
	int blockLength = histcount;

	assert(xRes*yRes % blockLength == 0);//Assert even division, otherwise kernel will crash.

	dim3 threads(blockLength);
	dim3 blocks((int)(ceil(float(xRes*yRes)/float(blockLength))));

	int sharedSize = histcount * sizeof(int);

	distanceHistogramKernel<<<blocks,threads,sharedSize>>>(dev_normalSegments, dev_planeProjectedDistanceMap, xRes, yRes, 
		dev_distanceHistograms[0], numMaxNormalSegments, histcount, histMinDist, histMaxDist);
}

#pragma endregion