#include "hip/hip_runtime.h"
#include "debug_rendering.h"


// Kernel that writes the depth image to the OpenGL PBO directly.
__global__ void sendDepthImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, DPixel* depthBuffer){

	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x) {

		// Cast to float for storage
		float depth = depthBuffer[i].depth;

		// Each thread writes one pixel location in the texture (textel)
		// Store depth in every component except alpha
		PBOpos[i].x = depth;
		PBOpos[i].y = depth;
		PBOpos[i].z = depth;
		PBOpos[i].w = 1.0f;
	}
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendColorImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, ColorPixel* colorBuffer){

	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x){

		glm::vec3 color;
		color.r = colorBuffer[i].r/255.0f;
		color.g = colorBuffer[i].g/255.0f;
		color.b = colorBuffer[i].b/255.0f;

		// Each thread writes one pixel location in the texture (textel)
		PBOpos[i].x = color.r;
		PBOpos[i].y = color.g;
		PBOpos[i].z = color.b;
		PBOpos[i].w = 1.0f;
	}
}


// Draws depth image buffer to the texture.
// Texture width and height must match the resolution of the depth image.
// Returns false if width or height does not match, true otherwise
__host__ void drawDepthImageBufferToPBO(float4* dev_PBOpos, DPixel* dev_depthImageBuffer, int texWidth, int texHeight)
{
	int tileSize = 16;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendDepthImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_depthImageBuffer);

}

// Draws color image buffer to the texture.
// Texture width and height must match the resolution of the color image.
// Returns false if width or height does not match, true otherwise
// dev_PBOpos must be a CUDA device pointer
__host__ void drawColorImageBufferToPBO(float4* dev_PBOpos, ColorPixel* dev_colorImageBuffer, int texWidth, int texHeight)
{
	int tileSize = 16;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendColorImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_colorImageBuffer);

}


__global__ void sendFloat3SOAToPBO(float4* pbo, float* x_src, float* y_src, float* z_src, float w,
								   int xRes, int yRes, int pboXRes, int pboYRes)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int i = (y * xRes) + x;
	int pboi = (y * pboXRes) + x;

	if(y < yRes && x < xRes){

		// Each thread writes one pixel location in the texture (textel)
		pbo[pboi].x = x_src[i];
		pbo[pboi].y = y_src[i];
		pbo[pboi].z = z_src[i];
		pbo[pboi].w = w;
	}
}


__global__ void clearPBOKernel(float4* pbo, int xRes, int yRes, float clearValue)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int i = (y * xRes) + x;

	if(y < yRes && x < xRes){

		// Each thread writes one pixel location in the texture (textel)
		// 
		float4 clear;
		clear.x = clearValue;
		clear.y = clearValue;
		clear.z = clearValue;
		clear.w = clearValue;
		pbo[i] = clear;
	}
}


__host__ void clearPBO(float4* pbo, int xRes, int yRes, float clearValue)
{
	int tileSize = 16;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(xRes)/float(tileSize)), 
		(int)ceil(float(yRes)/float(tileSize)));

	clearPBOKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(pbo, xRes, yRes, clearValue);

}

__host__ void drawVMaptoPBO(float4* pbo, Float3SOAPyramid vmap, int level, int xRes, int yRes)
{
	int tileSize = 16;

	if(level < NUM_PYRAMID_LEVELS)
	{
		int scaledXRes = xRes >> level;
		int scaledYRes = yRes >> level;

		dim3 threadsPerBlock(tileSize, tileSize);
		dim3 fullBlocksPerGrid((int)ceil(float(scaledXRes)/float(tileSize)), 
			(int)ceil(float(scaledYRes)/float(tileSize)));


		sendFloat3SOAToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(pbo, vmap.x[level], vmap.y[level], vmap.z[level],  1.0,
			scaledXRes, scaledYRes, xRes, yRes);
	}
}


__host__ void drawNMaptoPBO(float4* pbo, Float3SOAPyramid nmap, int level, int xRes, int yRes)
{
	int tileSize = 16;

	if(level < NUM_PYRAMID_LEVELS)
	{
		int scaledXRes = xRes >> level;
		int scaledYRes = yRes >> level;

		dim3 threadsPerBlock(tileSize, tileSize);
		dim3 fullBlocksPerGrid((int)ceil(float(scaledXRes)/float(tileSize)), 
			(int)ceil(float(scaledYRes)/float(tileSize)));


		sendFloat3SOAToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(pbo, nmap.x[level], nmap.y[level], nmap.z[level],  0.0,
			scaledXRes, scaledYRes, xRes, yRes);
	}
}



__host__ void drawRGBMaptoPBO(float4* pbo, RGBMapSOA rgbMap, int xRes, int yRes)
{
	int tileSize = 16;


	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(xRes)/float(tileSize)), 
		(int)ceil(float(yRes)/float(tileSize)));


	sendFloat3SOAToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(pbo, rgbMap.r, rgbMap.g, rgbMap.b,  1.0,
		xRes, yRes, xRes, yRes);

}