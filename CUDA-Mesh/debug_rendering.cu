#include "hip/hip_runtime.h"
#include "debug_rendering.h"


// Kernel that writes the depth image to the OpenGL PBO directly.
__global__ void sendDepthImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, DPixel* depthBuffer){

	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x) {

		// Cast to float for storage
		float depth = depthBuffer[i].depth;

		// Each thread writes one pixel location in the texture (textel)
		// Store depth in every component except alpha
		PBOpos[i].x = depth;
		PBOpos[i].y = depth;
		PBOpos[i].z = depth;
		PBOpos[i].w = 1.0f;
	}
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendColorImageBufferToPBO(float4* PBOpos, glm::vec2 resolution, ColorPixel* colorBuffer){

	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x){

		glm::vec3 color;
		color.r = colorBuffer[i].r/255.0f;
		color.g = colorBuffer[i].g/255.0f;
		color.b = colorBuffer[i].b/255.0f;

		// Each thread writes one pixel location in the texture (textel)
		PBOpos[i].x = color.r;
		PBOpos[i].y = color.g;
		PBOpos[i].z = color.b;
		PBOpos[i].w = 1.0f;
	}
}

__global__ void sendPCBToPBOs(float4* dptrPosition, float4* dptrColor, float4* dptrNormal, glm::vec2 resolution, PointCloud* dev_pcb)
{
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = (r * resolution.x) + c;

	if(r<resolution.y && c<resolution.x){

		PointCloud point = dev_pcb[i];

		// Each thread writes one pixel location in the texture (textel)

		dptrPosition[i].x = point.pos.x;
		dptrPosition[i].y = point.pos.y;
		dptrPosition[i].z = point.pos.z;
		dptrPosition[i].w = 1.0f;

		dptrColor[i].x = point.color.r;
		dptrColor[i].y = point.color.g;
		dptrColor[i].z = point.color.b;
		dptrColor[i].w = 1.0f;

		dptrNormal[i].x = point.normal.x;
		dptrNormal[i].y = point.normal.y;
		dptrNormal[i].z = point.normal.z;
		dptrNormal[i].w = 0.0f;
	}
}

// Draws depth image buffer to the texture.
// Texture width and height must match the resolution of the depth image.
// Returns false if width or height does not match, true otherwise
__host__ void drawDepthImageBufferToPBO(float4* dev_PBOpos, DPixel* dev_depthImageBuffer, int texWidth, int texHeight)
{
	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendDepthImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_depthImageBuffer);

}

// Draws color image buffer to the texture.
// Texture width and height must match the resolution of the color image.
// Returns false if width or height does not match, true otherwise
// dev_PBOpos must be a CUDA device pointer
__host__ void drawColorImageBufferToPBO(float4* dev_PBOpos, ColorPixel* dev_colorImageBuffer, int texWidth, int texHeight)
{
	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)));

	sendColorImageBufferToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(dev_PBOpos, glm::vec2(texWidth, texHeight), dev_colorImageBuffer);

}


// Renders various debug information about the 2D point cloud buffer to the texture.
// Texture width and height must match the resolution of the point cloud buffer.
// Returns false if width or height does not match, true otherwise
__host__ void drawPCBToPBO(float4* dptrPosition, float4* dptrColor, float4* dptrNormal, PointCloud* dev_pcb, int texWidth, int texHeight)
{
	int tileSize = 8;

	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid( (int)ceil(float(texWidth)/float(tileSize)), 
		(int)ceil(float(texHeight)/float(tileSize)) );

	sendPCBToPBOs<<<fullBlocksPerGrid, threadsPerBlock>>>(dptrPosition, dptrColor, dptrNormal, glm::vec2(texWidth, texHeight), dev_pcb);

}
