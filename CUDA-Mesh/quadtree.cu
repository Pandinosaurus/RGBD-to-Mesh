#include "hip/hip_runtime.h"
#include "quadtree.h"


__global__ void computeAABBsKernel(PlaneStats planeStats, int* planeInvIdMap, glm::vec3* tangents, glm::vec4* aabbs, 
								   int* planeCount, int maxPlanes,
								   Float3SOA positions, float* segmentProjectedSx, float* segmentProjectedSy, 
								   int* finalSegmentsBuffer, int xRes, int yRes)
{
	extern __shared__ int s_Mem[];
	int* s_InvMap = (int*) s_Mem;
	float* s_centroidX = (float*)(s_InvMap + maxPlanes);
	float* s_centroidY = s_centroidX + maxPlanes;
	float* s_centroidZ = s_centroidY + maxPlanes;
	glm::vec3* s_tangents = (glm::vec3*) (s_centroidZ + maxPlanes);
	glm::vec3* s_bitangents = s_tangents + maxPlanes;
	glm::vec4* s_aabb = (glm::vec4*)(s_bitangents + maxPlanes);

	int indexInBlock = threadIdx.x + threadIdx.y*blockDim.x;
	int imageX = threadIdx.x + blockDim.x*blockIdx.x;
	int imageY = threadIdx.y + blockDim.y*blockIdx.y;


	int numPlanes = planeCount[0];
	if(indexInBlock < maxPlanes)
	{
		s_InvMap[indexInBlock] = planeInvIdMap[indexInBlock];
		if(indexInBlock < numPlanes)
		{
			s_aabb[indexInBlock] = glm::vec4(0.0f);
			s_tangents[indexInBlock] = tangents[indexInBlock];
			s_centroidX[indexInBlock] = planeStats.centroids.x[indexInBlock];
			s_centroidY[indexInBlock] = planeStats.centroids.y[indexInBlock];
			s_centroidZ[indexInBlock] = planeStats.centroids.z[indexInBlock];
			//bitangent = norm cross tangent
			glm::vec3 norm(planeStats.norms.x[indexInBlock],planeStats.norms.y[indexInBlock],planeStats.norms.z[indexInBlock]);
			s_bitangents[indexInBlock] = glm::normalize(glm::cross(norm, s_tangents[indexInBlock]));
		}
	}
	__syncthreads();

	//Remap segments
	int segment = finalSegmentsBuffer[imageX + imageY*xRes];
	if(segment >= 0)
	{
		//Remap and writeback
		segment = s_InvMap[segment];
		finalSegmentsBuffer[imageX + imageY*xRes] = segment;

		//Compute Sx and Sy
		glm::vec3 dp = glm::vec3(positions.x[imageX + imageY*xRes], positions.y[imageX + imageY*xRes], positions.z[imageX + imageY*xRes])
			- glm::vec3(s_centroidX[segment], s_centroidY[segment], s_centroidZ[segment]);

		float sx = glm::dot(dp, s_bitangents[segment]);
		float sy = glm::dot(dp, s_tangents[segment]);
		segmentProjectedSx[imageX + imageY*xRes] = sx;
		segmentProjectedSy[imageX + imageY*xRes] = sy;

	}

}

__host__ void computeAABBs(PlaneStats planeStats, int* planeInvIdMap, glm::vec3* tangents, glm::vec4* aabbs, int* planeCount, int maxPlanes,
						   Float3SOA positions, float* segmentProjectedSx, float* segmentProjectedSy, 
						   int* finalSegmentsBuffer, int xRes, int yRes)
{
	int blockWidth = 32;
	int blockHeight = 8;

	assert(blockHeight*blockWidth >= maxPlanes);
	dim3 threads(blockWidth, blockHeight);
	dim3 blocks((int) ceil(xRes/float(blockWidth)), (int) ceil(yRes/float(blockHeight)));
	//plane map, tangent, bitangent, centroid and aabb of each plane loaded to shared memory.
	int sharedMem = maxPlanes*(sizeof(int) + sizeof(float)*3+sizeof(glm::vec3)*2 + sizeof(glm::vec4));

	computeAABBsKernel<<<blocks,threads,sharedMem>>>(planeStats, planeInvIdMap, tangents, aabbs, planeCount, maxPlanes,
		positions, segmentProjectedSx, segmentProjectedSy, 
		finalSegmentsBuffer, xRes, yRes);

}