#include "hip/hip_runtime.h"
#include "quadtree.h"

//Numthreads is assumed to be a power of two
__device__ void minmaxreduction(float* s_minSx, float* s_maxSx, float* s_minSy, float* s_maxSy, int indexInBlock, int nTotalThreads)
{
	int  thread2;
	float temp;

	while(nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.

		if (indexInBlock < halfPoint)
		{
			thread2 = indexInBlock + halfPoint;

			// Get the shared value stored by another thread
			temp = s_minSx[thread2];
			if (temp < s_minSx[indexInBlock]) 
				s_minSx[indexInBlock] = temp; 

			temp = s_minSy[thread2];
			if (temp < s_minSy[indexInBlock]) 
				s_minSy[indexInBlock] = temp; 

			temp = s_maxSx[thread2];
			if (temp > s_maxSx[indexInBlock]) 
				s_maxSx[indexInBlock] = temp; 

			temp = s_maxSy[thread2];
			if (temp > s_maxSy[indexInBlock]) 
				s_maxSy[indexInBlock] = temp; 
		}
		__syncthreads();

		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
}

__global__ void computeAABBsKernel(PlaneStats planeStats, int* planeInvIdMap, glm::vec3* tangents, glm::vec4* aabbs, 
								   int* planeCount, int maxPlanes,
								   Float3SOA positions, float* segmentProjectedSx, float* segmentProjectedSy, 
								   int* finalSegmentsBuffer, int xRes, int yRes)
{
	extern __shared__ int s_Mem[];
	int* s_InvMap = (int*) s_Mem;
	float* s_centroidX = (float*)(s_InvMap + maxPlanes);
	float* s_centroidY = s_centroidX + maxPlanes;
	float* s_centroidZ = s_centroidY + maxPlanes;
	glm::vec3* s_tangents = (glm::vec3*) (s_centroidZ + maxPlanes);
	glm::vec3* s_bitangents = s_tangents + maxPlanes;
	float* s_minSx = (float*)(s_bitangents + maxPlanes);
	float* s_minSy = (s_minSx + blockDim.x*blockDim.y);
	float* s_maxSx = (s_minSy + blockDim.x*blockDim.y);
	float* s_maxSy = (s_maxSx + blockDim.x*blockDim.y);

	int indexInBlock = threadIdx.x + threadIdx.y*blockDim.x;
	int imageX = threadIdx.x + blockDim.x*blockIdx.x;
	int imageY = threadIdx.y + blockDim.y*blockIdx.y;


	int numPlanes = planeCount[0];
	if(indexInBlock < maxPlanes)
	{
		s_InvMap[indexInBlock] = planeInvIdMap[indexInBlock];
		if(indexInBlock < numPlanes)
		{
			//s_aabb[indexInBlock] = glm::vec4(0.0f);
			s_tangents[indexInBlock] = tangents[indexInBlock];
			s_centroidX[indexInBlock] = planeStats.centroids.x[indexInBlock];
			s_centroidY[indexInBlock] = planeStats.centroids.y[indexInBlock];
			s_centroidZ[indexInBlock] = planeStats.centroids.z[indexInBlock];
			//bitangent = norm cross tangent
			glm::vec3 norm(planeStats.norms.x[indexInBlock],planeStats.norms.y[indexInBlock],planeStats.norms.z[indexInBlock]);
			s_bitangents[indexInBlock] = glm::normalize(glm::cross(norm, s_tangents[indexInBlock]));
		}
	}
	__syncthreads();

	//Remap segments
	int segment = finalSegmentsBuffer[imageX + imageY*xRes];
	float sx = 0;
	float sy = 0;
	if(segment >= 0)
	{
		//Remap and writeback
		segment = s_InvMap[segment];
		finalSegmentsBuffer[imageX + imageY*xRes] = segment;

		//Compute Sx and Sy
		glm::vec3 dp = glm::vec3(positions.x[imageX + imageY*xRes] - s_centroidX[segment], 
			positions.y[imageX + imageY*xRes] - s_centroidY[segment],
			positions.z[imageX + imageY*xRes] - s_centroidZ[segment]);

		sx = glm::dot(dp, s_bitangents[segment]);
		sy = glm::dot(dp, s_tangents[segment]);


	}
	//writeback
	segmentProjectedSx[imageX + imageY*xRes] = sx;
	segmentProjectedSy[imageX + imageY*xRes] = sy;

	__syncthreads();
	//Repurpose invmap sharedmem for segment flags

	if(indexInBlock < maxPlanes)
	{
		s_InvMap[indexInBlock] = 0;
	}

	__syncthreads();
	if(segment >= 0)//flag each segment that exists in this block
		s_InvMap[segment] = 1;

	for(int plane = 0; plane < numPlanes; ++plane)
	{
		if(s_InvMap[plane] > 0)
		{

			//Init minmax planes
			s_minSx[indexInBlock] = (segment == plane)?sx:0;
			s_maxSx[indexInBlock] = (segment == plane)?sx:0;
			s_minSy[indexInBlock] = (segment == plane)?sy:0;
			s_maxSy[indexInBlock] = (segment == plane)?sy:0;
			__syncthreads();
			minmaxreduction(s_minSx, s_maxSx, s_minSy, s_maxSy, indexInBlock, blockDim.x*blockDim.y);

		}
	}


}

__host__ void computeAABBs(PlaneStats planeStats, int* planeInvIdMap, glm::vec3* tangents, glm::vec4* aabbs, int* planeCount, int maxPlanes,
						   Float3SOA positions, float* segmentProjectedSx, float* segmentProjectedSy, 
						   int* finalSegmentsBuffer, int xRes, int yRes)
{
	int blockWidth = 32;
	int blockHeight = 8;

	assert(blockHeight*blockWidth >= maxPlanes);
	dim3 threads(blockWidth, blockHeight);
	dim3 blocks((int) ceil(xRes/float(blockWidth)), (int) ceil(yRes/float(blockHeight)));
	//plane map, tangent, bitangent, centroid and aabb of each plane loaded to shared memory.
	int sharedMem = maxPlanes*(sizeof(int) + sizeof(float)*3+sizeof(glm::vec3)*2) + blockWidth*blockHeight*4*sizeof(float);

	computeAABBsKernel<<<blocks,threads,sharedMem>>>(planeStats, planeInvIdMap, tangents, aabbs, planeCount, maxPlanes,
		positions, segmentProjectedSx, segmentProjectedSy, 
		finalSegmentsBuffer, xRes, yRes);

}