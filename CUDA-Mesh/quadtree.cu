#include "hip/hip_runtime.h"
#include "quadtree.h"

//R2 = R1*multFactor + R2;
__device__ void add_r1_to_r2(glm::mat3 &A, glm::vec3 &b, int r1, int r2, float multFactor)
{
	float tmp;

	if (r1 == r2) return;

	for(int i = 0; i < 3; ++i)
	{
		A[i][r2] = A[i][r2] + multFactor*A[i][r1];
	}

	b[r2] = b[r2] + multFactor*b[r1];

}

__device__ void swap_row(glm::mat3 &A, glm::vec3 &b, int r1, int r2)
{
	float tmp;

	if (r1 == r2) return;

#pragma unroll
	for(int i = 0; i < 3; ++i)
	{
		tmp = A[i][r1];
		A[i][r1] = A[i][r2];
		A[i][r2] = tmp;
	}

	tmp = b[r1];
	b[r1] = b[r2];
	b[r2] = tmp;

}

__device__ void row_mult(glm::mat3 &A, glm::vec3 &b, int r1, float mult)
{
#pragma unroll
	for(int i = 0; i < 3; ++i)
	{
		A[i][r1] = A[i][r1]*mult;
	}

	b[r1] *= mult;

}


#define APPROXZERO(a) (abs(a) < 0.000001f)
__device__ void makeNonZeroDiagonal(glm::mat3 &A,glm::vec3 b)
{
	int permute[3];

	if(		 !APPROXZERO(A[0][0]) && !APPROXZERO(A[1][1]) && !APPROXZERO(A[2][2]))
	{
		permute[0] = 0; permute[1] = 1; permute[2] = 2;
	}else if(!APPROXZERO(A[0][0]) && !APPROXZERO(A[1][2]) && !APPROXZERO(A[2][1]))
	{
		permute[0] = 0; permute[1] = 2; permute[2] = 1;
	}else if(!APPROXZERO(A[0][1]) && !APPROXZERO(A[1][0]) && !APPROXZERO(A[2][2]))
	{
		permute[0] = 1; permute[1] = 0; permute[2] = 2;
	}else if(!APPROXZERO(A[0][1]) && !APPROXZERO(A[1][2]) && !APPROXZERO(A[2][0]))
	{
		permute[0] = 1; permute[1] = 2; permute[2] = 0;
	}else if(!APPROXZERO(A[0][2]) && !APPROXZERO(A[1][0]) && !APPROXZERO(A[2][1]))
	{
		permute[0] = 2; permute[1] = 0; permute[2] = 1;
	}else if(!APPROXZERO(A[0][2]) && !APPROXZERO(A[1][1]) && !APPROXZERO(A[2][0]))
	{
		permute[0] = 2; permute[1] = 1; permute[2] = 0;
	}else{
		//ERROR
	}

	for(int i = 0; i < 3; ++i)
	{
		if(permute[i] > i)
		{
			swap_row(A,b, i, permute[i]);
			for(int j = 0; j < 3; ++j)
			{
				if(permute[j] == i)
				{
					permute[j] = permute[i];
					permute[i] = i;
					break;
				}
			}
		}
	}
}

__device__ glm::vec3 solveAbGaussian(glm::mat3 A, glm::vec3 b)
{
	//Make sure diagonals have non-zero entries
	if(abs(A[0][0]*A[1][1]*A[2][2]) < 0.000001f)
		makeNonZeroDiagonal(A,b);

	//Row echelon form
	for(int r = 0; r < 3; ++r)
	{
		float factor = 1.0f/A[r][r];
		row_mult(A,b,r,factor);
		for(int r2 = r+1; r2 < 3; ++r2)
		{
			if(abs(A[r][r2]) > 0.000001f)
			{
				//If A[r][r2] not zero yet, 
				//Need A[r][r2] + factor*A[r][r] == 0
				factor = -A[r][r2]/A[r][r]; 
				add_r1_to_r2(A,b,r,r2,factor);
			}
		}
	}

	//Matrix now upper triangular
	//Back substitute
	for(int r = 0; r < 3; ++r)
	{
		for(int c = r+1; c < 3; ++c)
		{
			if(abs(A[c][r]) > 0.000001f)
			{
				//element is non-zero. Backsubstitute
				//Need A[c][r] + factor*A[c][c] == 0
				float factor = -A[c][r]/A[c][c];
				add_r1_to_r2(A,b,c,r,factor);
			}
		}
	}
	return b;

}



//Numthreads is assumed to be a power of two
__device__ void minmaxreduction(float* s_minSx, float* s_maxSx, float* s_minSy, float* s_maxSy, int indexInBlock, int nTotalThreads)
{
	int  thread2;
	float temp;

	while(nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.

		if (indexInBlock < halfPoint)
		{
			thread2 = indexInBlock + halfPoint;

			// Get the shared value stored by another thread
			temp = s_minSx[thread2];
			if (temp < s_minSx[indexInBlock]) 
				s_minSx[indexInBlock] = temp; 

			temp = s_minSy[thread2];
			if (temp < s_minSy[indexInBlock]) 
				s_minSy[indexInBlock] = temp; 

			temp = s_maxSx[thread2];
			if (temp > s_maxSx[indexInBlock]) 
				s_maxSx[indexInBlock] = temp; 

			temp = s_maxSy[thread2];
			if (temp > s_maxSy[indexInBlock]) 
				s_maxSy[indexInBlock] = temp; 
		}
		__syncthreads();

		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
}

__global__ void computeAABBsKernel(PlaneStats* planeStats, int* planeInvIdMap, glm::vec4* aabbsBlockResults,
								   int* planeCount, int maxPlanes,
								   Float3SOA positions, float* segmentProjectedSx, float* segmentProjectedSy, 
								   int* finalSegmentsBuffer, int xRes, int yRes)
{
	extern __shared__ int s_Mem[];
	int* s_InvMap = (int*) s_Mem;
	float* s_centroidX = (float*)(s_InvMap + maxPlanes);
	float* s_centroidY = s_centroidX + maxPlanes;
	float* s_centroidZ = s_centroidY + maxPlanes;
	glm::vec3* s_tangents = (glm::vec3*) (s_centroidZ + maxPlanes);
	glm::vec3* s_bitangents = s_tangents + maxPlanes;
	float* s_minSx = (float*)(s_bitangents + maxPlanes);
	float* s_minSy = (s_minSx + blockDim.x*blockDim.y);
	float* s_maxSx = (s_minSy + blockDim.x*blockDim.y);
	float* s_maxSy = (s_maxSx + blockDim.x*blockDim.y);

	int indexInBlock = threadIdx.x + threadIdx.y*blockDim.x;
	int imageX = threadIdx.x + blockDim.x*blockIdx.x;
	int imageY = threadIdx.y + blockDim.y*blockIdx.y;


	int numPlanes = planeCount[0];
	if(indexInBlock < maxPlanes)
	{
		s_InvMap[indexInBlock] = planeInvIdMap[indexInBlock];
		if(indexInBlock < numPlanes)
		{
			//s_aabb[indexInBlock] = glm::vec4(0.0f);
			s_tangents[indexInBlock] = planeStats[indexInBlock].tangent;
			s_centroidX[indexInBlock] = planeStats[indexInBlock].centroid.x;
			s_centroidY[indexInBlock] = planeStats[indexInBlock].centroid.y;
			s_centroidZ[indexInBlock] = planeStats[indexInBlock].centroid.z;
			//bitangent = norm cross tangent
			glm::vec3 norm(planeStats[indexInBlock].norm.x,planeStats[indexInBlock].norm.y,planeStats[indexInBlock].norm.z);
			s_bitangents[indexInBlock] = glm::normalize(glm::cross(norm, s_tangents[indexInBlock]));
		}
	}
	__syncthreads();

	//Remap segments
	int segment = finalSegmentsBuffer[imageX + imageY*xRes];
	float sx = 0;
	float sy = 0;
	if(segment >= 0)
	{
		//Remap and writeback
		segment = s_InvMap[segment];
		finalSegmentsBuffer[imageX + imageY*xRes] = segment;

		//Compute Sx and Sy
		glm::vec3 dp = glm::vec3(positions.x[imageX + imageY*xRes] - s_centroidX[segment], 
			positions.y[imageX + imageY*xRes] - s_centroidY[segment],
			positions.z[imageX + imageY*xRes] - s_centroidZ[segment]);

		sx = glm::dot(dp, s_bitangents[segment]);
		sy = glm::dot(dp, s_tangents[segment]);


	}
	//writeback
	segmentProjectedSx[imageX + imageY*xRes] = sx;
	segmentProjectedSy[imageX + imageY*xRes] = sy;

	__syncthreads();
	//Repurpose invmap sharedmem for segment flags

	if(indexInBlock < maxPlanes)
	{
		s_InvMap[indexInBlock] = 0;
	}

	__syncthreads();
	if(segment >= 0)//flag each segment that exists in this block
		s_InvMap[segment] = 1;

	for(int plane = 0; plane < numPlanes; ++plane)
	{
		if(s_InvMap[plane] > 0)
		{

			//Init minmax planes
			s_minSx[indexInBlock] = (segment == plane)?sx:0;
			s_maxSx[indexInBlock] = (segment == plane)?sx:0;
			s_minSy[indexInBlock] = (segment == plane)?sy:0;
			s_maxSy[indexInBlock] = (segment == plane)?sy:0;
			__syncthreads();
			minmaxreduction(s_minSx, s_maxSx, s_minSy, s_maxSy, indexInBlock, blockDim.x*blockDim.y);
			//Threads already synced in function

			if(indexInBlock == 0)
			{
				aabbsBlockResults[(blockIdx.x + blockIdx.y*gridDim.x) + plane*gridDim.x*gridDim.y] 
					= glm::vec4(s_minSx[0], s_maxSx[0],s_minSy[0],s_maxSy[0]);
			}
		}else{
			if(indexInBlock == 0)
				aabbsBlockResults[(blockIdx.x + blockIdx.y*gridDim.x) + plane*gridDim.x*gridDim.y] = glm::vec4(0.0f);
		}
	}
}


__global__ void reduceAABBsKernel(PlaneStats* planeStats, glm::vec4* aabbsBlockResults, int numBlocks, int maxPlanes, int* planeCount)
{
	extern __shared__ float s_temp[];
	float* s_minSx = s_temp;
	float* s_minSy = (s_minSx + blockDim.x);
	float* s_maxSx = (s_minSy + blockDim.x);
	float* s_maxSy = (s_maxSx + blockDim.x);

	//two elements loaded per thread
	int i = threadIdx.x;
	int i2 = threadIdx.x + blockDim.x;

	int numPlanes = planeCount[0];
	for(int plane = 0; plane < numPlanes; ++plane)
	{
		glm::vec4 aabb1(0.0f);
		glm::vec4 aabb2(0.0f);
		if(i < numBlocks)
			aabb1 = aabbsBlockResults[i + plane*numBlocks];
		if(i2 < numBlocks)
			aabb2 = aabbsBlockResults[i2 + plane*numBlocks];

		s_minSx[i] = MIN(aabb1.x,aabb2.x);
		s_maxSx[i] = MAX(aabb1.y,aabb2.y);
		s_minSy[i] = MIN(aabb1.z,aabb2.z);
		s_maxSy[i] = MAX(aabb1.w,aabb2.w);

		__syncthreads();
		minmaxreduction(s_minSx, s_maxSx, s_minSy, s_maxSy, i, blockDim.x);

		if(threadIdx.x == 0)
			planeStats[plane].projParams.aabbMeters = glm::vec4(s_minSx[0], s_maxSx[0],s_minSy[0],s_maxSy[0]);
	}
}


__host__ __device__ int roundupnextpow2 (int x)
{
	if (x < 0)
		return 0;
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return x+1;
}

__host__ void computeAABBs(PlaneStats* planeStats, int* planeInvIdMap, glm::vec4* aabbsBlockResults,
						   int* planeCount, int maxPlanes,
						   Float3SOA positions, float* segmentProjectedSx, float* segmentProjectedSy, 
						   int* finalSegmentsBuffer, int xRes, int yRes)
{
	int blockWidth = AABB_COMPUTE_BLOCKWIDTH;
	int blockHeight = AABB_COMPUTE_BLOCKHEIGHT;

	assert(blockHeight*blockWidth >= maxPlanes);
	dim3 threads(blockWidth, blockHeight);
	dim3 blocks((int) ceil(xRes/float(blockWidth)), (int) ceil(yRes/float(blockHeight)));
	//plane map, tangent, bitangent, centroid and aabb of each plane loaded to shared memory.
	int sharedMem = maxPlanes*(sizeof(int) + sizeof(float)*3+sizeof(glm::vec3)*2) + blockWidth*blockHeight*4*sizeof(float);

	computeAABBsKernel<<<blocks,threads,sharedMem>>>(planeStats, planeInvIdMap, aabbsBlockResults, planeCount, maxPlanes,
		positions, segmentProjectedSx, segmentProjectedSy, 
		finalSegmentsBuffer, xRes, yRes);

	int numBlocks = blocks.x*blocks.y;
	int pow2Blocks = roundupnextpow2 (numBlocks) >> 1;//Next lowest power of two
	assert(pow2Blocks <= 1024);


	threads = dim3(pow2Blocks);
	blocks = dim3(1);
	sharedMem = 4*sizeof(float)*pow2Blocks;
	reduceAABBsKernel<<<blocks,threads,sharedMem>>>(planeStats, aabbsBlockResults, numBlocks, maxPlanes, planeCount);

}


__global__ void calculateProjectionDataKernel(rgbd::framework::Intrinsics intr, PlaneStats* planeStats,
											  int* planeCount, int maxTextureSize, int xRes, int yRes)
{
	glm::mat3 C(1.0f);

	int destWidth  = 0;
	int destHeight = 0;
	int maxRatio = 0;
	glm::vec4 aabb;
	if(threadIdx.x < planeCount[0])
	{
		//In range and valid plane.

		glm::vec3 tangent = planeStats[threadIdx.x].tangent;
		glm::vec3 normal = glm::vec3(planeStats[threadIdx.x].norm.x,planeStats[threadIdx.x].norm.y,planeStats[threadIdx.x].norm.z);
		glm::vec3 bitangent = glm::normalize(glm::cross(normal, tangent));

		glm::vec3 centroid = glm::vec3(planeStats[threadIdx.x].centroid.x,
			planeStats[threadIdx.x].centroid.y,
			planeStats[threadIdx.x].centroid.z);
		aabb = planeStats[threadIdx.x].projParams.aabbMeters;

		//Compute camera space coordinates (4 points in clockwise winding from viewpoint)
		/*   1----2
		*    |    |
		*    4----3
		*/
		glm::vec3 sp1 = (aabb.x*bitangent)+(aabb.z*tangent)+centroid;//UL, Sxmin,Symin
		glm::vec3 sp2 = (aabb.y*bitangent)+(aabb.z*tangent)+centroid;//UR, Sxmax,Symin
		glm::vec3 sp3 = (aabb.y*bitangent)+(aabb.w*tangent)+centroid;//LR, Sxmax,Symax
		glm::vec3 sp4 = (aabb.x*bitangent)+(aabb.w*tangent)+centroid;//LL, Sxmin,Symax

		//Compute screen space projections
		float su1 = sp1.x*intr.fx/sp1.z + intr.cx;
		float sv1 = sp1.y*intr.fy/sp1.z + intr.cy;
		float su2 = sp2.x*intr.fx/sp2.z + intr.cx;
		float sv2 = sp2.y*intr.fy/sp2.z + intr.cy;
		float su3 = sp3.x*intr.fx/sp3.z + intr.cx;
		float sv3 = sp3.y*intr.fy/sp3.z + intr.cy;
		float su4 = sp4.x*intr.fx/sp4.z + intr.cx;
		float sv4 = sp4.y*intr.fy/sp4.z + intr.cy;

		//Compute desired resolution.
		float sourceWidthMeters = aabb.y-aabb.x;
		float sourceHeightMeters = aabb.w-aabb.z;

		//Compute minimum resolution for complete data preservation
		float d12 = sqrtf((su1-su2)*(su1-su2)+(sv1-sv2)*(sv1-sv2));
		float d23 = sqrtf((su2-su3)*(su2-su3)+(sv2-sv3)*(sv2-sv3));
		float d34 = sqrtf((su3-su4)*(su3-su4)+(sv3-sv4)*(sv3-sv4));
		float d41 = sqrtf((su4-su1)*(su4-su1)+(sv4-sv1)*(sv4-sv1));
		float maxXRatio = MAX(d12,d34)/sourceWidthMeters;
		float maxYRatio = MAX(d23,d41)/sourceHeightMeters;

		maxRatio = ceil(MAX(maxXRatio,maxYRatio));
		maxRatio = roundupnextpow2(maxRatio);

		destWidth  = maxRatio * sourceWidthMeters;
		destHeight = maxRatio * sourceHeightMeters;

		//Make sure it fits. If not, then scale down
		if(destWidth > maxTextureSize || destHeight > maxTextureSize)
		{
			int scale = glm::max(ceil(destWidth/float(maxTextureSize)),ceil(destHeight/float(maxTextureSize)));
			scale = roundupnextpow2(scale);
			destWidth/=scale;
			destHeight/=scale;

		}

		//Compute A matrix (source points to basis vectors)
		glm::mat3 A = glm::mat3(su1,sv1,1,su2,sv2,1,su3,sv3,1);
		glm::vec3 b = glm::vec3(su4,sv4, 1);
		glm::vec3 x = glm::inverse(A)*b; 
		//mult each row i by xi
		for(int i = 0; i < 3; ++i)
		{
			A[i][0] *= x[i];
			A[i][1] *= x[i];
			A[i][2] *= x[i];
		}


		//Compute B matrix (dest points to basis vectors)
		glm::mat3 B = glm::mat3(0,0,1,
			destWidth,0,1,
			destWidth,destHeight,1);
		b = glm::vec3(0,destHeight, 1);

		x = glm::inverse(B)*b;
		//mult each row i by xi
		for(int i = 0; i < 3; ++i)
		{
			B[i][0] *= x[i];
			B[i][1] *= x[i];
			B[i][2] *= x[i];
		}

		C = A*glm::inverse(B);

		
	}

	planeStats[threadIdx.x].projParams.projectionMatrix = C;
	planeStats[threadIdx.x].projParams.aabbMeters = aabb;
	planeStats[threadIdx.x].projParams.destWidth = destWidth;
	planeStats[threadIdx.x].projParams.destHeight = destHeight;
	planeStats[threadIdx.x].projParams.textureResolution = maxRatio;
}


__host__ void calculateProjectionData(rgbd::framework::Intrinsics intr, PlaneStats* planeStats,
									  int* planeCount, int maxTextureSize, int maxPlanes, int xRes, int yRes)
{
	dim3 blocks(1);
	dim3 threads(maxPlanes);

	calculateProjectionDataKernel<<<blocks,threads>>>(intr, planeStats, planeCount, maxTextureSize, xRes, yRes);
}


__global__ void projectTexture(int segmentId, PlaneStats* dev_planeStats, 
							   Float4SOA destTexture, int destTextureSize, 
							   RGBMapSOA rgbMap, int* dev_finalSegmentsBuffer, float* dev_finalDistanceToPlaneBuffer,
							   int imageXRes, int imageYRes)
{
	int destX = blockIdx.x*blockDim.x+threadIdx.x;
	int destY = blockIdx.y*blockDim.y+threadIdx.y;

	if(destX < destTextureSize && destX < dev_planeStats->projParams.destWidth
		&& destY < destTextureSize && destY < dev_planeStats->projParams.destHeight)
	{
		float r = HIP_NAN_F;
		float g = HIP_NAN_F;
		float b = HIP_NAN_F;
		float dist = HIP_NAN_F;

		//Destination in range
		glm::mat3 Tds = dev_planeStats->projParams.projectionMatrix;

		glm::vec3 sourceCoords = Tds*glm::vec3(destX, destY, 1.0f);

		//Dehomogenization
		sourceCoords.x /= sourceCoords.z;
		sourceCoords.y /= sourceCoords.z;

		if(sourceCoords.x >= 0 && sourceCoords.x < imageXRes 
			&& sourceCoords.y >= 0 && sourceCoords.y < imageYRes )
		{
			//In source range
			int linIndex = int(sourceCoords.x) + int(sourceCoords.y)*imageXRes;
			if(segmentId == dev_finalSegmentsBuffer[linIndex]){
				r = rgbMap.r[linIndex];
				g = rgbMap.g[linIndex];
				b = rgbMap.b[linIndex];
				dist = dev_finalSegmentsBuffer[linIndex];
			}
		}

		destTexture.x[destX + destY*destTextureSize] = r;
		destTexture.y[destX + destY*destTextureSize] = g;
		destTexture.z[destX + destY*destTextureSize] = b;
		destTexture.w[destX + destY*destTextureSize] = dist;
	}

}


__host__ void projectTexture(int segmentId, PlaneStats* host_planeStats, PlaneStats* dev_planeStats,  
							 Float4SOA destTexture, int destTextureSize, 
							 RGBMapSOA rgbMap, int* dev_finalSegmentsBuffer, float* dev_finalDistanceToPlaneBuffer,
							 int imageXRes, int imageYRes)
{
	int tileSize = 16;

	dim3 threads(tileSize, tileSize);
	dim3 blocks((int)ceil(float(host_planeStats->projParams.destWidth)/float(tileSize)),
		(int)ceil(float(host_planeStats->projParams.destHeight)/float(tileSize)));

	projectTexture<<<blocks,threads>>>(segmentId, dev_planeStats, destTexture, destTextureSize, 
		rgbMap, dev_finalSegmentsBuffer, dev_finalDistanceToPlaneBuffer, imageXRes, imageYRes);
}


__global__ void quadtreeDecimationKernel1(int actualWidth, int actualHeight, Float4SOA planarTexture, int* quadTreeAssemblyBuffer,
										  int textureBufferSize)
{
	extern __shared__ int s_tile[];

	//======================Load==========================
	//Global index
	int gx = threadIdx.x + blockDim.x*blockIdx.x;
	int gy = threadIdx.y + blockDim.y*blockIdx.y;
	int s_index = threadIdx.x + threadIdx.y*(blockDim.x+1);
	int indexInBlock = threadIdx.x + threadIdx.y*blockDim.x;
	//Load shared memory
	//load core. If in range and texture buffer has valid pixel at this location, load 0. Else, load -1;
	int val = -1;
	if(gx < actualWidth && gy < actualHeight)
	{
		float pixelContents = planarTexture.x[gx+gy*textureBufferSize];
		if(pixelContents == pixelContents)
		{
			val = 0;//Pixel is valid point. save
		}
	}
	s_tile[s_index] = val;

	//Load apron
	if(indexInBlock < (blockDim.x*2+1))//first 33 threads load remaining apron
	{
		if(indexInBlock < blockDim.x)//first 16 load bottom
		{
			gx = indexInBlock  + blockDim.x*blockIdx.x;
			gy = blockDim.y*(blockIdx.y+1);//first row of next block
			s_index = indexInBlock + (blockDim.y*(blockDim.x+1));
		}else if(indexInBlock < blockDim.x*2){//next 16 load right apron
			gx = blockDim.x*(blockIdx.x+1);//First column of next block
			gy = blockDim.y*blockIdx.y + (indexInBlock % blockDim.x);//indexInBlock % blockDim.x is y position in block
			s_index = blockDim.x + ((indexInBlock % blockDim.x)*(blockDim.x+1));
		}else{
			//load the corner
			gx = blockDim.x*(blockIdx.x+1);
			gy = blockDim.y*(blockIdx.y+1);
			s_index = blockDim.x + blockDim.y*(blockDim.x+1);
		}

		val = -1;
		if(gx < actualWidth && gy < actualHeight)
		{
			float pixelContents = planarTexture.x[gx+gy*textureBufferSize];
			if(pixelContents == pixelContents)
			{
				val = 0;//Pixel is valid point. save
			}

		}
		s_tile[s_index] = val;
	}
	__syncthreads();

	//====================Reduction=========================


	//Step == 0 is special case. need to initialize baseline quads
	bool merge = false;
	if(s_tile[threadIdx.x+threadIdx.y*(blockDim.x+1)] == 0)
	{
		//Check neighbors. If all neighbors right down and right-down diagonal are 0, set to 1.
		if(		s_tile[(threadIdx.x+1)	+	(threadIdx.y)  *(blockDim.x+1)] == 0
			&&	s_tile[(threadIdx.x  )	+	(threadIdx.y+1)*(blockDim.x+1)] == 0
			&&	s_tile[(threadIdx.x+1)	+	(threadIdx.y+1)*(blockDim.x+1)] == 0)
		{
			merge = true;
		}
	}

	__syncthreads();
	if(merge)
	{
		s_tile[threadIdx.x+threadIdx.y*(blockDim.x+1)] = 1;
	}

	__syncthreads();


	//Loop for remaining steps
	for(int step = 1; step < blockDim.x; step <<= 1)
	{
		if((threadIdx.x % (step*2)) == 0 && (threadIdx.y % (step*2)) == 0)
		{
			//Corner points only.
			if(	s_tile[(threadIdx.x)	+	(threadIdx.y)  *(blockDim.x+1)] == step
				&&  s_tile[(threadIdx.x+step)	+	(threadIdx.y)	  *(blockDim.x+1)] == step
				&&	s_tile[(threadIdx.x		)	+	(threadIdx.y+step)*(blockDim.x+1)] == step
				&&	s_tile[(threadIdx.x+step)	+	(threadIdx.y+step)*(blockDim.x+1)] == step)
			{
				//Upgrade degree of this point
				s_tile[(threadIdx.x)	+	(threadIdx.y)  *(blockDim.x+1)] *= 2;

				//Clear definitely removed points
				s_tile[(threadIdx.x+step)	+	(threadIdx.y)  *(blockDim.x+1)] = -1;
				s_tile[(threadIdx.x		)	+	(threadIdx.y+step)*(blockDim.x+1)] = -1;
				s_tile[(threadIdx.x+step)	+	(threadIdx.y+step)*(blockDim.x+1)] = -1;

			}
		}
		__syncthreads();

	}

	//====================Writeback=========================
	//writeback core.
	gx = threadIdx.x + blockDim.x*blockIdx.x;
	gy = threadIdx.y + blockDim.y*blockIdx.y;
	s_index = threadIdx.x + threadIdx.y*(blockDim.x+1);
	quadTreeAssemblyBuffer[gx+gy*textureBufferSize] = s_tile[s_index];

	//no need to writeback apron

}

__global__ void quadtreeDecimationKernel2(int actualWidth, int actualHeight, int* quadTreeAssemblyBuffer, int textureBufferSize)
{
	extern __shared__ int s_tile[];

	int scaleMultiplier = blockDim.x;

	//======================Load==========================
	//Global index (scaled by multiplier)
	int gx = scaleMultiplier*(threadIdx.x + blockDim.x*blockIdx.x);
	int gy = scaleMultiplier*(threadIdx.y + blockDim.y*blockIdx.y);
	int s_index = threadIdx.x + threadIdx.y*(blockDim.x+1);
	int indexInBlock = threadIdx.x + threadIdx.y*blockDim.x;
	//Load shared memory
	//load core. If in range and texture buffer has valid pixel at this location, load 0. Else, load -1;
	int val = -1;
	if(gx < actualWidth && gy < actualHeight)
	{
		val = quadTreeAssemblyBuffer[gx+gy*textureBufferSize];
	}
	s_tile[s_index] = val;

	//Load apron
	if(indexInBlock < (blockDim.x*2+1))//first 33 threads load remaining apron
	{
		if(indexInBlock < blockDim.x)//first 16 load bottom
		{
			gx = indexInBlock  + blockDim.x*blockIdx.x;
			gy = blockDim.y*(blockIdx.y+1);//first row of next block
			s_index = indexInBlock + (blockDim.y*(blockDim.x+1));
		}else if(indexInBlock < blockDim.x*2){//next 16 load right apron
			gx = blockDim.x*(blockIdx.x+1);//First column of next block
			gy = blockDim.y*blockIdx.y + (indexInBlock % blockDim.x);//indexInBlock % blockDim.x is y position in block
			s_index = blockDim.x + ((indexInBlock % blockDim.x)*(blockDim.x+1));
		}else{
			//load the corner
			gx = blockDim.x*(blockIdx.x+1);
			gy = blockDim.y*(blockIdx.y+1);
			s_index = blockDim.x + blockDim.y*(blockDim.x+1);
		}
		gx *= scaleMultiplier;
		gy *= scaleMultiplier;

		val = -1;
		if(gx < actualWidth && gy < actualHeight)
		{
			val = quadTreeAssemblyBuffer[gx+gy*textureBufferSize];
		}
		s_tile[s_index] = val;
	}
	__syncthreads();

	//====================Reduction=========================


	//Step == 0 is special case. need to initialize baseline quads
	bool merge = false;
	if(s_tile[threadIdx.x+threadIdx.y*(blockDim.x+1)] == 0)
	{
		//Check neighbors. If all neighbors right down and right-down diagonal are 0, set to 1.
		if(		s_tile[(threadIdx.x+1)	+	(threadIdx.y)  *(blockDim.x+1)] == 0
			&&	s_tile[(threadIdx.x  )	+	(threadIdx.y+1)*(blockDim.x+1)] == 0
			&&	s_tile[(threadIdx.x+1)	+	(threadIdx.y+1)*(blockDim.x+1)] == 0)
		{
			merge = true;
		}
	}

	__syncthreads();
	if(merge)
	{
		s_tile[threadIdx.x+threadIdx.y*(blockDim.x+1)] = 1;
	}

	__syncthreads();


	//Loop for remaining steps
	for(int step = 1; step < blockDim.x; step <<= 1)
	{
		if((threadIdx.x % (step*2)) == 0 && (threadIdx.y % (step*2)) == 0)
		{
			//Corner points only.
			if(	s_tile[(threadIdx.x)	+	(threadIdx.y)  *(blockDim.x+1)] == step*scaleMultiplier
				&&  s_tile[(threadIdx.x+step)	+	(threadIdx.y)	  *(blockDim.x+1)] == step*scaleMultiplier
				&&	s_tile[(threadIdx.x		)	+	(threadIdx.y+step)*(blockDim.x+1)] == step*scaleMultiplier
				&&	s_tile[(threadIdx.x+step)	+	(threadIdx.y+step)*(blockDim.x+1)] == step*scaleMultiplier)
			{
				//Upgrade degree of this point
				s_tile[(threadIdx.x)	+	(threadIdx.y)  *(blockDim.x+1)] *= 2;

				//Clear definitely removed points
				s_tile[(threadIdx.x+step)	+	(threadIdx.y)  *(blockDim.x+1)] = -1;
				s_tile[(threadIdx.x		)	+	(threadIdx.y+step)*(blockDim.x+1)] = -1;
				s_tile[(threadIdx.x+step)	+	(threadIdx.y+step)*(blockDim.x+1)] = -1;

			}
		}
		__syncthreads();

	}

	//====================Writeback=========================
	//writeback core.
	gx = scaleMultiplier*(threadIdx.x + blockDim.x*blockIdx.x);
	gy = scaleMultiplier*(threadIdx.y + blockDim.y*blockIdx.y);
	s_index = threadIdx.x + threadIdx.y*(blockDim.x+1);
	quadTreeAssemblyBuffer[gx+gy*textureBufferSize] = s_tile[s_index];

	//no need to writeback apron

}

__host__ void quadtreeDecimation(int actualWidth, int actualHeight, Float4SOA planarTexture, int* quadTreeAssemblyBuffer,
								 int textureBufferSize)
{
	//do two simplification passes. Max quadtree size will therefore be 2*tileSize
	int tileSize = 16;

	//Pass one, parallel by pixel
	dim3 threads(tileSize, tileSize);
	dim3 blocks((int)ceil(actualWidth/float(tileSize)),
		(int)ceil(actualHeight/float(tileSize)));
	int sharedSize = (tileSize+1)*(tileSize+1)*sizeof(int);
	quadtreeDecimationKernel1<<<blocks,threads,sharedSize>>>(actualWidth, actualHeight, planarTexture, quadTreeAssemblyBuffer, textureBufferSize);

	blocks = dim3((int)ceil(actualWidth/float(tileSize*tileSize)),
		(int)ceil(actualHeight/float(tileSize*tileSize)));
	quadtreeDecimationKernel2<<<blocks,threads,sharedSize>>>(actualWidth, actualHeight, quadTreeAssemblyBuffer, textureBufferSize);

}



#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_SIZE 65535

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5 

#define NO_BANK_CONFLICTS


#ifdef NO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n)    \
	(((n) >> (2 * LOG_NUM_BANKS)))  
#else
#define CONFLICT_FREE_OFFSET(a)    (0)  
#endif


__global__ void quadTreeExclusiveScanKernel(int width, int* input, int* output,  int bufferStride, int* blockResults)
{
	extern __shared__ float temp[];

	//Offset pointers to this block's row. Avoids the need for more complex indexing
	input += bufferStride*blockIdx.x;
	output += bufferStride*blockIdx.x;

	//Now each row is working with it's own row like a normal exclusive scan of an array length width.
	int index = threadIdx.x;
	int offset = 1;
	int n = 2*blockDim.x;//get actual temp padding

	int ai = index;
	int bi = index + n/2;
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	//Bounds checking, load shared mem
	temp[ai+bankOffsetA] = (ai < width)?input[ai]:0;
	temp[bi+bankOffsetB] = (bi < width)?input[bi]:0;
	//Negative vertecies are to be cleared
	if(temp[ai+bankOffsetA] < 0)
		temp[ai+bankOffsetA] = 0;
	if(temp[bi+bankOffsetB] < 0)
		temp[bi+bankOffsetB] = 0;


	//Reduction step
	for (int d = n>>1; d > 0; d >>= 1)                  
	{   
		__syncthreads();  //Make sure previous step has completed
		if (index < d)  
		{
			int ai2 = offset*(2*index+1)-1;  
			int bi2 = offset*(2*index+2)-1;  
			ai2 += CONFLICT_FREE_OFFSET(ai2);
			bi2 += CONFLICT_FREE_OFFSET(bi2);

			temp[bi2] += temp[ai2];
		}  
		offset *= 2;  //Adjust offset
	}

	//Reduction complete

	//Clear last element
	if(index == 0)
	{
		blockResults[blockIdx.x] = temp[(n-1)+CONFLICT_FREE_OFFSET(n-1)];
		temp[(n-1)+CONFLICT_FREE_OFFSET(n-1)] = 0;
	}

	//Sweep down
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
	{  
		offset >>= 1;  
		__syncthreads();  //wait for previous step to finish
		if (index < d)                       
		{  
			int ai2 = offset*(2*index+1)-1;  
			int bi2 = offset*(2*index+2)-1;  
			ai2 += CONFLICT_FREE_OFFSET(ai2);
			bi2 += CONFLICT_FREE_OFFSET(bi2);

			//Swap
			float t = temp[ai2];  
			temp[ai2] = temp[bi2];  
			temp[bi2] += t;   
		}  
	}  

	//Sweep complete
	__syncthreads();

	//Writeback
	if(ai < width)
		output[ai] = temp[ai+bankOffsetA];
	if(bi < width)
		output[bi] = temp[bi+bankOffsetB];

}


__global__ void blockResultsExclusiveScanKernel(int* blockResults, int numBlocks, int* totalSumOut)
{
	extern __shared__ float temp[];

	//Now each row is working with it's own row like a normal exclusive scan of an array length width.
	int index = threadIdx.x;
	int offset = 1;
	int n = 2*blockDim.x;//get actual temp padding

	int ai = index;
	int bi = index + n/2;
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	//Bounds checking, load shared mem
	temp[ai+bankOffsetA] = (ai < numBlocks)?blockResults[ai]:0;
	temp[bi+bankOffsetB] = (bi < numBlocks)?blockResults[bi]:0;

	//Reduction step
	for (int d = n>>1; d > 0; d >>= 1)                  
	{   
		__syncthreads();  //Make sure previous step has completed
		if (index < d)  
		{
			int ai2 = offset*(2*index+1)-1;  
			int bi2 = offset*(2*index+2)-1;  
			ai2 += CONFLICT_FREE_OFFSET(ai2);
			bi2 += CONFLICT_FREE_OFFSET(bi2);

			temp[bi2] += temp[ai2];
		}  
		offset *= 2;  //Adjust offset
	}

	//Reduction complete
	//Clear last element
	if(index == 0)
	{
		totalSumOut[0] = temp[(n-1)+CONFLICT_FREE_OFFSET(n-1)];
		temp[(n-1)+CONFLICT_FREE_OFFSET(n-1)] = 0;
	}

	//Sweep down
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
	{  
		offset >>= 1;  
		__syncthreads();  //wait for previous step to finish
		if (index < d)                       
		{  
			int ai2 = offset*(2*index+1)-1;  
			int bi2 = offset*(2*index+2)-1;  
			ai2 += CONFLICT_FREE_OFFSET(ai2);
			bi2 += CONFLICT_FREE_OFFSET(bi2);

			//Swap
			float t = temp[ai2];  
			temp[ai2] = temp[bi2];  
			temp[bi2] += t;   
		}  
	}  

	//Sweep complete
	__syncthreads();

	//Writeback
	if(ai < numBlocks)
		blockResults[ai] = temp[ai+bankOffsetA];
	if(bi < numBlocks)
		blockResults[bi] = temp[bi+bankOffsetB];


}



__global__ void reintegrateResultsKernel(int actualWidth, int textureBufferSize, 
										 int* quadTreeScanResults,  int* blockResults)
{
	int pixelX = threadIdx.x;
	int pixelY = blockIdx.x;

	if(pixelX < actualWidth)
	{
		quadTreeScanResults[pixelX + pixelY*textureBufferSize] += blockResults[pixelY];
	}
}


__global__ void scatterResultsKernel(glm::vec4 aabbMeters, int actualWidth, int actualHeight, 
									 int finalTextureWidth, int finalTextureHeight, int textureBufferSize, 
									 int* quadTreeAssemblyBuffer,  int* quadTreeScanResults,  
									 int* blockResults,  int* indexBuffer, float4* vertexBuffer)
{
	int pixelX = threadIdx.x;
	int pixelY = blockIdx.x;

	if(pixelX < actualWidth)
	{
		int degree = quadTreeAssemblyBuffer[pixelX + pixelY*textureBufferSize];//Load vertex degree

		//Only continue if this is a used vertex in the quadtree
		if(degree >= 0)
		{
			int vertNum = quadTreeScanResults[pixelX + pixelY*textureBufferSize];

			//Compute vertex info.
			float textureU = float(pixelX)/float(finalTextureWidth);
			float textureV = float(pixelY)/float(finalTextureHeight);

			//pixelX*(Sxmax-Sxmin)/actualWidth + Sxmin;
			float posX = (pixelX*(aabbMeters.y-aabbMeters.x))/float(actualWidth) + aabbMeters.x;
			//pixelY*(Symax-Symin)/actualHeight + Symin;
			float posY = (pixelY*(aabbMeters.w-aabbMeters.z))/float(actualHeight) + aabbMeters.z;

			float4 vertex;
			vertex.x = posX;
			vertex.y = posY;
			vertex.z = textureU;
			vertex.w = textureV;
			vertexBuffer[vertNum] = vertex;


			//Generate mesh
			// Quad configuration:
			// 0-1
			// |/|
			// 2-3
			// Index order: 0 - 1 - 2, 2 - 1 -3
			//Already loaded vertnum for 0
			int vertNum0 = 0;
			int vertNum1 = 0;
			int vertNum2 = 0;
			int vertNum3 = 0;

			//If degree greater than 0, assemble quad
			if(degree > 0)
			{
				//garunteed to be in range by nature of quadtree degree
				vertNum0 = vertNum;
				vertNum1 = quadTreeScanResults[(pixelX+degree) + (pixelY)*textureBufferSize];
				vertNum2 = quadTreeScanResults[(pixelX) + (pixelY+degree)*textureBufferSize];
				vertNum3 = quadTreeScanResults[(pixelX+degree) + (pixelY+degree)*textureBufferSize];
			}

			//Always fill buffer
			// Index order: 0 - 1 - 2, 2 - 1 -3
			int offset = vertNum*6;
			indexBuffer[offset+0] = vertNum0;
			indexBuffer[offset+1] = vertNum1;
			indexBuffer[offset+2] = vertNum2;
			indexBuffer[offset+3] = vertNum2;
			indexBuffer[offset+4] = vertNum1;
			indexBuffer[offset+5] = vertNum3;

		}
	}
}

__global__ void reshapeTextureKernel(int actualWidth, int actualHeight, int finalTextureWidth, int finalTextureHeight, int textureBufferSize, 
		Float4SOA planarTexture, float4* finalTexture)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int destIndex = x + y * finalTextureWidth;
	int sourceIndex = x + y * textureBufferSize;

	float4 textureValue = {HIP_NAN_F,HIP_NAN_F,HIP_NAN_F,HIP_NAN_F};

	if(x < actualWidth && y < actualHeight)
	{
		textureValue.x = planarTexture.x[sourceIndex];
		textureValue.y = planarTexture.y[sourceIndex];
		textureValue.z = planarTexture.z[sourceIndex];
		textureValue.w = planarTexture.w[sourceIndex];
	}

	
	finalTexture[destIndex] = textureValue;
}


__host__ void quadtreeMeshGeneration(glm::vec4 aabbMeters, int actualWidth, int actualHeight, int* quadTreeAssemblyBuffer,
									 int* quadTreeScanResults, int textureBufferSize, int* blockResults, int blockResultsBufferSize,
									 int* indexBuffer, float4* vertexBuffer, int* compactCount, int* host_compactCount, int outputBufferSize,
									 int finalTextureWidth, int finalTextureHeight, Float4SOA planarTexture, float4* finalTexture)
{
	int blockSize = roundupnextpow2(actualWidth);
	int numBlocks = actualHeight;
	dim3 threads(blockSize >> 1);//2 elements per thread
	dim3 blocks(numBlocks);
	int sharedCount = (blockSize+2)*sizeof(int);


	//Make sure size constraints aren't violated
	assert(blocks.x <= blockResultsBufferSize);
	assert(blockResultsBufferSize <= blockSize);

	//Scan blocks
	quadTreeExclusiveScanKernel<<<blocks,threads,sharedCount>>>(actualWidth, quadTreeAssemblyBuffer, 
		quadTreeScanResults, textureBufferSize, blockResults);

	//Scan block results
	int pow2 = roundupnextpow2(numBlocks);
	threads = dim3(pow2>>1);
	blocks = dim3(1);
	assert(pow2 <= blockResultsBufferSize);

	sharedCount = (pow2 + 2)*sizeof(int);
	blockResultsExclusiveScanKernel<<<blocks,threads,sharedCount>>>(blockResults, numBlocks, compactCount);

	hipMemcpy(host_compactCount, compactCount, sizeof(int), hipMemcpyDeviceToHost);

	//Reintegrate
	//Also scatter (generate meshes and vertecies in the process)
	threads = dim3(actualWidth);
	blocks = dim3(numBlocks);
	reintegrateResultsKernel<<<blocks,threads>>>(actualWidth, textureBufferSize, quadTreeScanResults, blockResults);

	assert(finalTextureWidth <= textureBufferSize);
	assert(finalTextureHeight <= textureBufferSize);

	scatterResultsKernel<<<blocks,threads>>>(aabbMeters, actualWidth, actualHeight, finalTextureWidth, finalTextureHeight, textureBufferSize, 
		quadTreeAssemblyBuffer, quadTreeScanResults, blockResults, indexBuffer, vertexBuffer);


	//Reshape texture to aligned memory
	threads = dim3(finalTextureWidth);
	blocks = dim3(finalTextureHeight);
	reshapeTextureKernel<<<blocks,threads>>>(actualWidth, actualHeight, finalTextureWidth, finalTextureHeight, textureBufferSize, 
		planarTexture, finalTexture);
}